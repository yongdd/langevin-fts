#include <thrust/reduce.h>
#include "CudaCommon.h"

class TestCudaCub
{

public:

    const int M{32*32*32};

    // Variables for cub reduction sum
    size_t temp_storage_bytes;
    double *d_temp_storage;
    double *d_array;
    double *d_array_sum;

    TestCudaCub()
    {
        gpu_error_check(hipMalloc((void**)&d_array, sizeof(double)*M));
        gpu_error_check(hipMalloc((void**)&d_array_sum, sizeof(double)*1));

        temp_storage_bytes = 0;
        d_temp_storage = nullptr;

        std::cout << "temp_storage_bytes before: " << temp_storage_bytes << std::endl; 
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_array, d_array_sum, M);
        std::cout << "temp_storage_bytes after: " << temp_storage_bytes << std::endl;
    };

    ~TestCudaCub()
    {
        hipFree(d_array);
        hipFree(d_array_sum);
        hipFree(d_temp_storage);
    };

};
int main()
{
    try{
        TestCudaCub testcub = TestCudaCub();
        if (testcub.temp_storage_bytes == 0) {
            std::cout << "ERROR: temp_storage_bytes is 0" << std::endl;
            return -1;
        } else {
            std::cout << "SUCCESS: temp_storage_bytes: " << testcub.temp_storage_bytes << std::endl;
            return 0;
        }
    }
    catch(std::exception& exc)
    {
        std::cout << exc.what() << std::endl;
        return -1;
    }
}