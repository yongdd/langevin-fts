#include <thrust/reduce.h>
#include "CudaCommon.h"

int main()
{
    try{
        const int M{32*32*32};
        // const int N_GPUS = CudaCommon::get_instance().get_n_gpus();

        double array[32*32*32];
        for (int i=0; i<M; i++)
            array[i] = i;

        // Variables for cub reduction sum
        size_t temp_storage_bytes;
        double *d_temp_storage;
        double *d_array;
        double *d_array_sum;
        double array_sum;

        array_sum = 0.0;
        gpu_error_check(hipMalloc((void**)&d_array, sizeof(double)*M));
        gpu_error_check(hipMalloc((void**)&d_array_sum, sizeof(double)*1));
        gpu_error_check(hipMemcpy(d_array, array, sizeof(double)*M, hipMemcpyHostToDevice));

        // It seems that hipcub::DeviceReduce::Sum changes temp_storage_bytes[gpu],
        // If d_temp_storage[gpu]=nullptr and temp_storage_bytes[gpu]=0.
        d_temp_storage = nullptr;
        temp_storage_bytes = 0;
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_array, d_array_sum, M);
        gpu_error_check(hipMalloc(&d_temp_storage, temp_storage_bytes));

        gpu_error_check(hipMemcpy(&array_sum, d_array_sum, sizeof(double), hipMemcpyDeviceToHost));
        std::cout << "gpu, array_sum 0: " << 0 << ", " << array_sum << std::endl;

        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_array, d_array_sum, M);
        gpu_error_check(hipMemcpy(&array_sum, d_array_sum, sizeof(double), hipMemcpyDeviceToHost));
        std::cout << "gpu, array_sum 1: " << 0 << ", " << array_sum << std::endl;

        hipFree(d_array);
        hipFree(d_array_sum);
        hipFree(d_temp_storage);
        return 0;
    }
    catch(std::exception& exc)
    {
        std::cout << exc.what() << std::endl;
        return -1;
    }
}