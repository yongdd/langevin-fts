#include "hip/hip_runtime.h"
#include <algorithm>
#include "CudaCommon.h"
#include "PinnedCircularBuffer.h"

template <typename T>
PinnedCircularBuffer<T>::PinnedCircularBuffer(int length, int width)
{
    this->length = length;
    this->width = width;
    this->start = 0;
    this->n_items = 0;

    elems = new T*[length];
    for(int i=0; i<length; i++)
    {
        gpu_error_check(hipHostMalloc((void**)&elems[i], sizeof(T)*width));
        for(int j=0; j<width; j++)
            elems[i][j] = 0.0;
    }
}
template <typename T>
PinnedCircularBuffer<T>::~PinnedCircularBuffer()
{
    for(int i=0; i<length; i++)
        hipHostFree(elems[i]);
    delete[] elems;
}
template <typename T>
void PinnedCircularBuffer<T>::reset()
{
    start = 0;
    n_items = 0;
}
template <typename T>
void PinnedCircularBuffer<T>::insert(T* new_arr)
{
    int i = (start+n_items)%length;
    for(int m=0; m<width; m++){
        elems[i][m] = new_arr[m];
    }
    if (n_items == length)
        start = (start+1)%length;
    n_items = std::min(n_items+1, length);
}
template <typename T>
T* PinnedCircularBuffer<T>::get_array(int n)
{
    int i = (start+n_items-n-1+length)%length;
    return elems[i];
}
// double* PinnedCircularBuffer<T>::operator[] (int n)
// {
//     int i = (start+n_items-n-1+length)%length;
//     return elems[i];
// }
// double PinnedCircularBuffer<T>::get(int n, int m)
// {
//     int i = (start+n_items-n-1+length)%length;
//     return elems[i][m];
// }


// Explicit template instantiation
template class PinnedCircularBuffer<double>;
template class PinnedCircularBuffer<std::complex<double>>;