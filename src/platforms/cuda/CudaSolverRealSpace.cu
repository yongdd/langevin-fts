#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include "CudaSolverRealSpace.h"

CudaSolverRealSpace::CudaSolverRealSpace(
    ComputationBox<double>* cb,
    Molecules *molecules,
    int n_streams,
    hipStream_t streams[MAX_STREAMS][2],
    bool reduce_gpu_memory_usage)
{
    try{
        this->cb = cb;
        this->molecules = molecules;
        this->chain_model = molecules->get_model_name();
        this->n_streams = n_streams;
        this->reduce_gpu_memory_usage = reduce_gpu_memory_usage;

        if(molecules->get_model_name() != "continuous")
            throw_with_line_number("Real-space method only support 'continuous' chain model.");     

        const int M = cb->get_total_grid();
        const int DIM = cb->get_dim();
        this->dim = DIM;
        std::vector<int> nx(DIM);
        if (DIM == 3)
            nx = {cb->get_nx(0), cb->get_nx(1), cb->get_nx(2)};
        else if (DIM == 2)
            nx = {cb->get_nx(0), cb->get_nx(1), 1};
        else if (DIM == 1)
            nx = {cb->get_nx(0), 1, 1};

        // Copy streams
        for(int i=0; i<n_streams; i++)
        {
            this->streams[i][0] = streams[i][0];
            this->streams[i][1] = streams[i][1];
        }

        // Create boltz_bond, boltz_bond_half, exp_dw, and exp_dw_half
        for(const auto& item: molecules->get_bond_lengths())
        {
            std::string monomer_type = item.first;
            d_exp_dw     [monomer_type] = nullptr;
            d_exp_dw_half[monomer_type] = nullptr;

            gpu_error_check(hipMalloc((void**)&d_exp_dw     [monomer_type], sizeof(double)*M));
            gpu_error_check(hipMalloc((void**)&d_exp_dw_half[monomer_type], sizeof(double)*M));

            d_xl[monomer_type] = nullptr;
            d_xd[monomer_type] = nullptr;
            d_xh[monomer_type] = nullptr;
            gpu_error_check(hipMalloc((void**)&d_xl[monomer_type], sizeof(double)*nx[0]));
            gpu_error_check(hipMalloc((void**)&d_xd[monomer_type], sizeof(double)*nx[0]));
            gpu_error_check(hipMalloc((void**)&d_xh[monomer_type], sizeof(double)*nx[0]));

            d_yl[monomer_type] = nullptr;
            d_yd[monomer_type] = nullptr;
            d_yh[monomer_type] = nullptr;
            gpu_error_check(hipMalloc((void**)&d_yl[monomer_type], sizeof(double)*nx[1]));
            gpu_error_check(hipMalloc((void**)&d_yd[monomer_type], sizeof(double)*nx[1]));
            gpu_error_check(hipMalloc((void**)&d_yh[monomer_type], sizeof(double)*nx[1]));

            d_zl[monomer_type] = nullptr;
            d_zd[monomer_type] = nullptr;
            d_zh[monomer_type] = nullptr;
            gpu_error_check(hipMalloc((void**)&d_zl[monomer_type], sizeof(double)*nx[2]));
            gpu_error_check(hipMalloc((void**)&d_zd[monomer_type], sizeof(double)*nx[2]));
            gpu_error_check(hipMalloc((void**)&d_zh[monomer_type], sizeof(double)*nx[2]));
        }

        if(DIM == 3)
        {
            for(int i=0; i<n_streams; i++)
            {
                gpu_error_check(hipMalloc((void**)&d_q_star[i], sizeof(double)*M));
                gpu_error_check(hipMalloc((void**)&d_q_dstar[i], sizeof(double)*M));
                gpu_error_check(hipMalloc((void**)&d_c_star[i], sizeof(double)*M));
                gpu_error_check(hipMalloc((void**)&d_q_sparse[i], sizeof(double)*M));
                gpu_error_check(hipMalloc((void**)&d_temp[i], sizeof(double)*M));
            }

            int offset_xy[nx[0]*nx[1]];
            int offset_yz[nx[1]*nx[2]];
            int offset_xz[nx[0]*nx[2]];
            int count;

            count = 0;
            for(int i=0;i<nx[0];i++)
                for(int j=0;j<nx[1];j++)
                    offset_xy[count++] = i*nx[1]*nx[2] + j*nx[2];

            count = 0;
            for(int j=0;j<nx[1];j++)
                for(int k=0;k<nx[2];k++)
                    offset_yz[count++] = j*nx[2] + k;

            count = 0;
            for(int i=0;i<nx[0];i++)
                for(int k=0;k<nx[2];k++)
                    offset_xz[count++] = i*nx[1]*nx[2] + k;

            gpu_error_check(hipMalloc((void**)&d_offset_xy, sizeof(int)*nx[0]*nx[1]));
            gpu_error_check(hipMalloc((void**)&d_offset_yz, sizeof(int)*nx[1]*nx[2]));
            gpu_error_check(hipMalloc((void**)&d_offset_xz, sizeof(int)*nx[0]*nx[2]));

            gpu_error_check(hipMemcpy(d_offset_xy, offset_xy, sizeof(int)*nx[0]*nx[1], hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_offset_yz, offset_yz, sizeof(int)*nx[1]*nx[2], hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_offset_xz, offset_xz, sizeof(int)*nx[0]*nx[2], hipMemcpyHostToDevice));
        }
        else if(DIM == 2)
        {
            for(int i=0; i<n_streams; i++)
            {
                gpu_error_check(hipMalloc((void**)&d_q_star[i], sizeof(double)*M));
                gpu_error_check(hipMalloc((void**)&d_c_star[i], sizeof(double)*M));
                gpu_error_check(hipMalloc((void**)&d_q_sparse[i], sizeof(double)*M));
                gpu_error_check(hipMalloc((void**)&d_temp[i], sizeof(double)*M));
            }

            int offset_x[nx[0]];
            int offset_y[nx[1]];

            for(int i=0;i<nx[0];i++)
                offset_x[i] = i*nx[1];

            for(int j=0;j<nx[1];j++)
                offset_y[j] = j;

            gpu_error_check(hipMalloc((void**)&d_offset_x, sizeof(int)*nx[0]));
            gpu_error_check(hipMalloc((void**)&d_offset_y, sizeof(int)*nx[1]));

            gpu_error_check(hipMemcpy(d_offset_x, offset_x, sizeof(int)*nx[0], hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_offset_y, offset_y, sizeof(int)*nx[1], hipMemcpyHostToDevice));
        }
        else if(DIM == 1)
        {
            for(int i=0; i<n_streams; i++)
            {
                gpu_error_check(hipMalloc((void**)&d_q_star[i], sizeof(double)*M));
                gpu_error_check(hipMalloc((void**)&d_c_star[i], sizeof(double)*M));
                gpu_error_check(hipMalloc((void**)&d_q_sparse[i], sizeof(double)*M));
            }

            gpu_error_check(hipMalloc((void**)&d_offset, sizeof(int)));
            gpu_error_check(hipMemset(d_offset, 0, sizeof(int)));
        }

        update_laplacian_operator();
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
CudaSolverRealSpace::~CudaSolverRealSpace()
{
    const int DIM = this->dim;

    for(const auto& item: d_exp_dw)
        hipFree(item.second);
    for(const auto& item: d_exp_dw_half)
        hipFree(item.second);
    
    for(const auto& item: d_xl)
        hipFree(item.second);
    for(const auto& item: d_xd)
        hipFree(item.second);
    for(const auto& item: d_xh)
        hipFree(item.second);

    for(const auto& item: d_yl)
        hipFree(item.second);
    for(const auto& item: d_yd)
        hipFree(item.second);
    for(const auto& item: d_yh)
        hipFree(item.second);

    for(const auto& item: d_zl)
        hipFree(item.second);
    for(const auto& item: d_zd)
        hipFree(item.second);
    for(const auto& item: d_zh)
        hipFree(item.second);

    if(DIM == 3)
    {
        for(int i=0; i<n_streams; i++)
        {
            hipFree(d_q_star[i]);
            hipFree(d_q_dstar[i]);
            hipFree(d_c_star[i]);
            hipFree(d_q_sparse[i]);
            hipFree(d_temp[i]);
        }
        hipFree(d_offset_xy);
        hipFree(d_offset_yz);
        hipFree(d_offset_xz);
    }
    else if(DIM == 2)
    {
        for(int i=0; i<n_streams; i++)
        {
            hipFree(d_q_star[i]);
            hipFree(d_c_star[i]);
            hipFree(d_q_sparse[i]);
            hipFree(d_temp[i]);
        }
        hipFree(d_offset_x);
        hipFree(d_offset_y);
    }
    else if(DIM == 1)
    {
        for(int i=0; i<n_streams; i++)
        {
            hipFree(d_q_star[i]);
            hipFree(d_c_star[i]);
            hipFree(d_q_sparse[i]);
        }
        hipFree(d_offset);
    }
}
void CudaSolverRealSpace::update_laplacian_operator()
{
    try
    {
        const int M = this->cb->get_total_grid();
        const int DIM = this->cb->get_dim();
        std::vector<int> nx(DIM);
        if (DIM == 3)
            nx = {this->cb->get_nx(0), this->cb->get_nx(1), this->cb->get_nx(2)};
        else if (DIM == 2)
            nx = {this->cb->get_nx(0), this->cb->get_nx(1), 1};
        else if (DIM == 1)
            nx = {this->cb->get_nx(0), 1, 1};

        double xl[nx[0]], xd[nx[0]], xh[nx[0]];
        double yl[nx[1]], yd[nx[1]], yh[nx[1]];
        double zl[nx[2]], zd[nx[2]], zh[nx[2]];

        for(const auto& item: this->molecules->get_bond_lengths())
        {
            std::string monomer_type = item.first;
            double bond_length_sq = item.second*item.second;

            FiniteDifference::get_laplacian_matrix(
                this->cb->get_boundary_conditions(),
                this->cb->get_nx(), this->cb->get_dx(),
                xl, xd, xh,
                yl, yd, yh,
                zl, zd, zh,
                bond_length_sq, this->molecules->get_ds());

            gpu_error_check(hipMemcpy(d_xl[monomer_type], xl, sizeof(double)*nx[0], hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_xd[monomer_type], xd, sizeof(double)*nx[0], hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_xh[monomer_type], xh, sizeof(double)*nx[0], hipMemcpyHostToDevice));

            gpu_error_check(hipMemcpy(d_yl[monomer_type], yl, sizeof(double)*nx[1], hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_yd[monomer_type], yd, sizeof(double)*nx[1], hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_yh[monomer_type], yh, sizeof(double)*nx[1], hipMemcpyHostToDevice));

            gpu_error_check(hipMemcpy(d_zl[monomer_type], zl, sizeof(double)*nx[2], hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_zd[monomer_type], zd, sizeof(double)*nx[2], hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_zh[monomer_type], zh, sizeof(double)*nx[2], hipMemcpyHostToDevice));
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaSolverRealSpace::update_dw(std::string device, std::map<std::string, const double*> w_input)
{
    try{
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = this->cb->get_total_grid();
        const double ds = this->molecules->get_ds();

        for(const auto& item: w_input)
        {
            if( d_exp_dw.find(item.first) == d_exp_dw.end())
                throw_with_line_number("monomer_type \"" + item.first + "\" is not in d_exp_dw.");     
        }

        hipMemcpyKind cudaMemcpyInputToDevice;
        if (device == "gpu")
            cudaMemcpyInputToDevice = hipMemcpyDeviceToDevice;
        else if(device == "cpu")
            cudaMemcpyInputToDevice = hipMemcpyHostToDevice;
        else
        {
            throw_with_line_number("Invalid device \"" + device + "\".");
        }

        // Compute exp_dw and exp_dw_half
        for(const auto& item: w_input)
        {
            std::string monomer_type = item.first;
            const double *w = item.second;

            // Copy field configurations from host to device
            gpu_error_check(hipMemcpyAsync(
                d_exp_dw     [monomer_type], w,      
                sizeof(double)*M, cudaMemcpyInputToDevice));
            gpu_error_check(hipMemcpyAsync(
                d_exp_dw_half[monomer_type], w,
                sizeof(double)*M, cudaMemcpyInputToDevice));

            // Compute d_exp_dw and d_exp_dw_half
            ker_exp<<<N_BLOCKS, N_THREADS>>>
                ((double*) d_exp_dw[monomer_type],
                 (double*) d_exp_dw[monomer_type],      1.0, -0.50*ds, M);
            ker_exp<<<N_BLOCKS, N_THREADS>>>
                ((double*) d_exp_dw_half[monomer_type],
                 (double*) d_exp_dw_half[monomer_type], 1.0, -0.25*ds, M);

            gpu_error_check(hipDeviceSynchronize());
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaSolverRealSpace::advance_propagator(
    const int STREAM,
    double *d_q_in, double *d_q_out,
    std::string monomer_type, double *d_q_mask) 
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = this->cb->get_total_grid();
        const int DIM = this->cb->get_dim();

        double *_d_exp_dw = d_exp_dw[monomer_type];

        // Evaluate exp(-w*ds/2) in real space
        ker_multi<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(d_q_out, d_q_in, _d_exp_dw, 1.0, M);

        if(DIM == 3)           // input, output
            advance_propagator_3d(this->cb->get_boundary_conditions(), STREAM, d_q_out, d_q_out, monomer_type);
        else if(DIM == 2)
            advance_propagator_2d(this->cb->get_boundary_conditions(), STREAM, d_q_out, d_q_out, monomer_type);
        else if(DIM ==1 )
            advance_propagator_1d(this->cb->get_boundary_conditions(), STREAM, d_q_out, d_q_out, monomer_type);

        // Evaluate exp(-w*ds/2) in real space
        ker_multi<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(d_q_out, d_q_out, _d_exp_dw, 1.0, M);

        // Multiply mask
        if (d_q_mask != nullptr)
            ker_multi<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(d_q_out, d_q_out, d_q_mask, 1.0, M);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaSolverRealSpace::advance_propagator_3d(
    std::vector<BoundaryCondition> bc,
    const int STREAM,
    double *d_q_in, double *d_q_out, std::string monomer_type)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();
        const int M = this->cb->get_total_grid();
        const std::vector<int> nx = this->cb->get_nx();

        double *_d_xl = d_xl[monomer_type];
        double *_d_xd = d_xd[monomer_type];
        double *_d_xh = d_xh[monomer_type];

        double *_d_yl = d_yl[monomer_type];
        double *_d_yd = d_yd[monomer_type];
        double *_d_yh = d_yh[monomer_type];

        double *_d_zl = d_zl[monomer_type];
        double *_d_zd = d_zd[monomer_type];
        double *_d_zh = d_zh[monomer_type];

        // Calculate q_star
        compute_crank_3d_step_1<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
            bc[0], bc[1], bc[2], bc[3], bc[4], bc[5], 
            _d_xl, _d_xd, _d_xh, nx[0],
            _d_yl, _d_yd, _d_yh, nx[1],
            _d_zl, _d_zd, _d_zh, nx[2],
            d_temp[STREAM], d_q_in, M);

        if (bc[0] == BoundaryCondition::PERIODIC)
        {
            tridiagonal_periodic<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
                _d_xl, _d_xd, _d_xh,
                d_c_star[STREAM], d_q_sparse[STREAM], d_temp[STREAM], d_q_star[STREAM],
                d_offset_yz, nx[1]*nx[2], nx[1]*nx[2], nx[0]);
        }
        else
        {
            tridiagonal<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
                _d_xl, _d_xd, _d_xh,
                d_c_star[STREAM], d_temp[STREAM], d_q_star[STREAM],
                d_offset_yz, nx[1]*nx[2], nx[1]*nx[2], nx[0]);
        }

        // Calculate q_dstar
        compute_crank_3d_step_2<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
            bc[2], bc[3], 
            _d_yl, _d_yd, _d_yh, nx[1], nx[2],
            d_temp[STREAM], d_q_star[STREAM], d_q_in, M);

        if (bc[2] == BoundaryCondition::PERIODIC)
        {
            tridiagonal_periodic<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
                _d_yl, _d_yd, _d_yh,
                d_c_star[STREAM], d_q_sparse[STREAM], d_temp[STREAM], d_q_dstar[STREAM],
                d_offset_xz, nx[0]*nx[2], nx[2], nx[1]);
        }
        else
        {
            tridiagonal<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
                _d_yl, _d_yd, _d_yh,
                d_c_star[STREAM], d_temp[STREAM], d_q_dstar[STREAM],
                d_offset_xz, nx[0]*nx[2], nx[2], nx[1]);
        }

        // Calculate q^(n+1)
        compute_crank_3d_step_3<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
            bc[4], bc[5], 
            _d_zl, _d_zd, _d_zh, nx[1], nx[2],
            d_temp[STREAM], d_q_dstar[STREAM], d_q_in, M);

        if (bc[4] == BoundaryCondition::PERIODIC)
        {
            tridiagonal_periodic<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
                _d_zl, _d_zd, _d_zh,
                d_c_star[STREAM], d_q_sparse[STREAM], d_temp[STREAM], d_q_out,
                d_offset_xy, nx[0]*nx[1], 1, nx[2]);
        }
        else
        {
            tridiagonal<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
                _d_zl, _d_zd, _d_zh,
                d_c_star[STREAM], d_temp[STREAM], d_q_out,
                d_offset_xy, nx[0]*nx[1], 1, nx[2]);
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaSolverRealSpace::advance_propagator_2d(
    std::vector<BoundaryCondition> bc,
    const int STREAM,
    double *d_q_in, double *d_q_out, std::string monomer_type)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();
        const int M = this->cb->get_total_grid();
        const std::vector<int> nx = this->cb->get_nx();

        double *_d_xl = d_xl[monomer_type];
        double *_d_xd = d_xd[monomer_type];
        double *_d_xh = d_xh[monomer_type];

        double *_d_yl = d_yl[monomer_type];
        double *_d_yd = d_yd[monomer_type];
        double *_d_yh = d_yh[monomer_type];

        // Calculate q_star
        compute_crank_2d_step_1<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
            bc[0], bc[1], bc[2], bc[3],
            _d_xl, _d_xd, _d_xh, nx[0],
            _d_yl, _d_yd, _d_yh, nx[1],
            d_temp[STREAM], d_q_in, M);

        // gpu_error_check(hipMemcpy(d_q_out, d_q_star, sizeof(double)*M, hipMemcpyDeviceToDevice));

        if (bc[0] == BoundaryCondition::PERIODIC)
        {
            tridiagonal_periodic<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
                _d_xl, _d_xd, _d_xh,
                d_c_star[STREAM], d_q_sparse[STREAM], d_temp[STREAM], d_q_star[STREAM],
                d_offset_y, nx[1], nx[1], nx[0]);
        }
        else
        {
            tridiagonal<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
                _d_xl, _d_xd, _d_xh,
                d_c_star[STREAM], d_temp[STREAM], d_q_star[STREAM],
                d_offset_y, nx[1], nx[1], nx[0]);
        }

        // Calculate q_dstar
        compute_crank_2d_step_2<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
            bc[2], bc[3],
            _d_yl, _d_yd, _d_yh, nx[1],
            d_temp[STREAM], d_q_star[STREAM], d_q_in, M);

        if (bc[2] == BoundaryCondition::PERIODIC)
        {
            tridiagonal_periodic<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
                _d_yl, _d_yd, _d_yh,
                d_c_star[STREAM], d_q_sparse[STREAM], d_temp[STREAM], d_q_out,
                d_offset_x, nx[0], 1, nx[1]);
        }
        else
        {
            tridiagonal<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
                _d_yl, _d_yd, _d_yh,
                d_c_star[STREAM], d_temp[STREAM], d_q_out,
                d_offset_x, nx[0], 1, nx[1]);
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaSolverRealSpace::advance_propagator_1d(
    std::vector<BoundaryCondition> bc,
    const int STREAM,
    double *d_q_in, double *d_q_out, std::string monomer_type)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();
        const int M = this->cb->get_total_grid();
        const std::vector<int> nx = this->cb->get_nx();

        double *_d_xl = d_xl[monomer_type];
        double *_d_xd = d_xd[monomer_type];
        double *_d_xh = d_xh[monomer_type];

        compute_crank_1d<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
            bc[0], bc[1],
            _d_xl, _d_xd, _d_xh,
            d_q_star[STREAM], d_q_in, nx[0]);

        if (bc[0] == BoundaryCondition::PERIODIC)
            tridiagonal_periodic<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
                _d_xl, _d_xd, _d_xh,
                d_c_star[STREAM], d_q_sparse[STREAM], d_q_star[STREAM], d_q_out,
                d_offset, 1, 1, nx[0]);
        else
            tridiagonal<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
                _d_xl, _d_xd, _d_xh,
                d_c_star[STREAM], d_q_star[STREAM], d_q_out, d_offset, 1, 1, nx[0]);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaSolverRealSpace::compute_single_segment_stress(
        const int STREAM,
        double *d_q_pair, double *d_segment_stress,
        std::string monomer_type, bool is_half_bond_length)
{
    try
    {
        throw_with_line_number("Currently, the real-space method does not support stress computation.");
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}

__device__ int d_max_of_two(int x, int y)
{
   return (x > y) ? x : y;
}
__device__ int d_min_of_two(int x, int y)
{
   return (x < y) ? x : y;
}

__global__ void compute_crank_3d_step_1(
    BoundaryCondition bc_xl, BoundaryCondition bc_xh,
    BoundaryCondition bc_yl, BoundaryCondition bc_yh,
    BoundaryCondition bc_zl, BoundaryCondition bc_zh,
    const double *d_xl, const double *d_xd, const double *d_xh, const int I,
    const double *d_yl, const double *d_yd, const double *d_yh, const int J,
    const double *d_zl, const double *d_zd, const double *d_zh, const int K,
    double *d_q_out, const double *d_q_in, const int M)
{
    int im, ip, jm, jp, km, kp;

    int n = blockIdx.x * blockDim.x + threadIdx.x;
    while(n < M)
    {
        int i = n / (J*K);
        int j = (n-i*J*K) / K;
        int k = n % K;

        if (bc_xl == BoundaryCondition::PERIODIC)
            im = (I+i-1) % I;
        else
            im = d_max_of_two(0,i-1);
        if (bc_xh == BoundaryCondition::PERIODIC)
            ip = (i+1) % I;
        else
            ip = d_min_of_two(I-1,i+1);

        if (bc_yl == BoundaryCondition::PERIODIC)
            jm = (J+j-1) % J;
        else
            jm = d_max_of_two(0,j-1);
        if (bc_yh == BoundaryCondition::PERIODIC)
            jp = (j+1) % J;
        else
            jp = d_min_of_two(J-1,j+1);

        if (bc_zl == BoundaryCondition::PERIODIC)
            km = (K+k-1) % K;
        else
            km = d_max_of_two(0,k-1);
        if (bc_zh == BoundaryCondition::PERIODIC)
            kp = (k+1) % K;
        else
            kp = d_min_of_two(K-1,k+1);

        int im_j_k = im*J*K + j*K + k;
        int ip_j_k = ip*J*K + j*K + k;
        int i_jm_k = i*J*K + jm*K + k;
        int i_jp_k = i*J*K + jp*K + k;
        int i_j_km = i*J*K + j*K + km;
        int i_j_kp = i*J*K + j*K + kp;

        d_q_out[n] = 2.0*((3.0-0.5*d_xd[i]-d_yd[j]-d_zd[k])*d_q_in[n]
                - d_zl[k]*d_q_in[i_j_km] - d_zh[k]*d_q_in[i_j_kp]
                - d_yl[j]*d_q_in[i_jm_k] - d_yh[j]*d_q_in[i_jp_k])
                - d_xl[i]*d_q_in[im_j_k] - d_xh[i]*d_q_in[ip_j_k];

        n += blockDim.x * gridDim.x;
    }
}

__global__ void compute_crank_3d_step_2(
    BoundaryCondition bc_yl, BoundaryCondition bc_yh,
    const double *d_yl, const double *d_yd, const double *d_yh, const int J, const int K,
    double *d_q_out, const double *d_q_star, const double *d_q_in, const int M)
{
    int jm, jp;

    int n = blockIdx.x * blockDim.x + threadIdx.x;
    while(n < M)
    {
        int i = n / (J*K);
        int j = (n-i*J*K) / K;
        int k = n % K;

        if (bc_yl == BoundaryCondition::PERIODIC)
            jm = (J+j-1) % J;
        else
            jm = d_max_of_two(0,j-1);
        if (bc_yh == BoundaryCondition::PERIODIC)
            jp = (j+1) % J;
        else
            jp = d_min_of_two(J-1,j+1);

        int i_jm_k = i*J*K + jm*K + k;
        int i_jp_k = i*J*K + jp*K + k;

        d_q_out[n] = d_q_star[n] + (d_yd[j]-1.0)*d_q_in[n]
            + d_yl[j]*d_q_in[i_jm_k] + d_yh[j]*d_q_in[i_jp_k];

        n += blockDim.x * gridDim.x;
    }
}

__global__ void compute_crank_3d_step_3(
    BoundaryCondition bc_zl, BoundaryCondition bc_zh,
    const double *d_zl, const double *d_zd, const double *d_zh, const int J, const int K,
    double *d_q_out, const double *d_q_dstar, const double *d_q_in, const int M)
{
    int km, kp;

    int n = blockIdx.x * blockDim.x + threadIdx.x;
    while(n < M)
    {
        int i = n / (J*K);
        int j = (n-i*J*K) / K;
        int k = n % K;

        if (bc_zl == BoundaryCondition::PERIODIC)
            km = (K+k-1) % K;
        else
            km = d_max_of_two(0,k-1);
        if (bc_zh == BoundaryCondition::PERIODIC)
            kp = (k+1) % K;
        else
            kp = d_min_of_two(K-1,k+1);

        int i_j_km = i*J*K + j*K + km;
        int i_j_kp = i*J*K + j*K + kp;

        d_q_out[n] = d_q_dstar[n] + (d_zd[k]-1.0)*d_q_in[n]
            + d_zl[k]*d_q_in[i_j_km] + d_zh[k]*d_q_in[i_j_kp];

        n += blockDim.x * gridDim.x;
    }
}

__global__ void compute_crank_2d_step_1(
    BoundaryCondition bc_xl, BoundaryCondition bc_xh,
    BoundaryCondition bc_yl, BoundaryCondition bc_yh,
    const double *d_xl, const double *d_xd, const double *d_xh, const int I,
    const double *d_yl, const double *d_yd, const double *d_yh, const int J,
    double *d_q_out, const double *d_q_in, const int M)
{
    int im, ip, jm, jp;

    int n = blockIdx.x * blockDim.x + threadIdx.x;
    while(n < M)
    {
        int i = n / J;
        int j = n % J;

        if (bc_xl == BoundaryCondition::PERIODIC)
            im = (I+i-1) % I;
        else
            im = d_max_of_two(0,i-1);
        if (bc_xh == BoundaryCondition::PERIODIC)
            ip = (i+1) % I;
        else
            ip = d_min_of_two(I-1,i+1);

        if (bc_yl == BoundaryCondition::PERIODIC)
            jm = (J+j-1) % J;
        else
            jm = d_max_of_two(0,j-1);
        if (bc_yh == BoundaryCondition::PERIODIC)
            jp = (j+1) % J;
        else
            jp = d_min_of_two(J-1,j+1);

        int i_jm = i*J + jm;
        int i_jp = i*J + jp;
        int im_j = im*J + j;
        int ip_j = ip*J + j;

        d_q_out[n] = 2.0*((2.0-0.5*d_xd[i]-d_yd[j])*d_q_in[n]
                   - d_yl[j]*d_q_in[i_jm] - d_yh[j]*d_q_in[i_jp])
                   - d_xl[i]*d_q_in[im_j] - d_xh[i]*d_q_in[ip_j];

        n += blockDim.x * gridDim.x;
    }
}

__global__ void compute_crank_2d_step_2(
    BoundaryCondition bc_yl, BoundaryCondition bc_yh,
    const double *d_yl, const double *d_yd, const double *d_yh, const int J,
    double *d_q_out, const double *d_q_star, const double *d_q_in, const int M)
{
    int jm, jp;

    int n = blockIdx.x * blockDim.x + threadIdx.x;
    while(n < M)
    {
        int i = n/J;
        int j = n%J;

        if (bc_yl == BoundaryCondition::PERIODIC)
            jm = (J+j-1) % J;
        else
            jm = d_max_of_two(0,j-1);
        if (bc_yh == BoundaryCondition::PERIODIC)
            jp = (j+1) % J;
        else
            jp = d_min_of_two(J-1,j+1);

        int i_jm = i*J + jm;
        int i_jp = i*J + jp;

        d_q_out[n] = d_q_star[n] + (d_yd[j]-1.0)*d_q_in[n]
            + d_yl[j]*d_q_in[i_jm] + d_yh[j]*d_q_in[i_jp];

        n += blockDim.x * gridDim.x;
    }
}

__global__ void compute_crank_1d(
    BoundaryCondition bc_xl, BoundaryCondition bc_xh,
    const double *d_xl, const double *d_xd, const double *d_xh,
    double *d_q_out, const double *d_q_in, const int M)
{
    int im, ip;

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while(i < M)
    {
        if (bc_xl == BoundaryCondition::PERIODIC)
            im = (M+i-1) % M;
        else
            im = d_max_of_two(0,i-1);
        if (bc_xh == BoundaryCondition::PERIODIC)
            ip = (i+1) % M;
        else
            ip = d_min_of_two(M-1,i+1);

        // B part of Ax=B matrix equation
        d_q_out[i] = (2.0-d_xd[i])*d_q_in[i] - d_xl[i]*d_q_in[im] - d_xh[i]*d_q_in[ip];

        i += blockDim.x * gridDim.x;
    }
}

// This method solves CX=Y, where C is a tridiagonal matrix 
__global__ void tridiagonal(
    const double *d_xl, const double *d_xd, const double *d_xh,
    double *d_c_star,  const double *d_d, double *d_x,
    const int *d_offset, const int REPEAT,
    const int INTERVAL, const int M)
{
    // d_xl: a
    // d_xd: b
    // d_xh: c

    double temp;

    int n = blockIdx.x * blockDim.x + threadIdx.x;
    while (n < REPEAT)
    {
        const double *_d_d = &d_d[d_offset[n]];
        double       *_d_x = &d_x[d_offset[n]];
        double  *_d_c_star = &d_c_star[d_offset[n]];

        // Forward sweep
        temp = d_xd[0];
        _d_c_star[0] = d_xh[0]/d_xd[0];
        _d_x[0] = _d_d[0]/d_xd[0];

        for(int i=1; i<M; i++)
        {
            _d_c_star[(i-1)*INTERVAL] = d_xh[i-1]/temp;
            temp = d_xd[i]-d_xl[i]*_d_c_star[(i-1)*INTERVAL];
            _d_x[i*INTERVAL] = (_d_d[i*INTERVAL]-d_xl[i]*_d_x[(i-1)*INTERVAL])/temp;
        }

        // Backward substitution
        for(int i=M-2;i>=0; i--)
            _d_x[i*INTERVAL] = _d_x[i*INTERVAL] - _d_c_star[i*INTERVAL]*_d_x[(i+1)*INTERVAL];
        
        n += blockDim.x * gridDim.x;
    }
}

// This method solves CX=Y, where C is a near-tridiagonal matrix with periodic boundary condition
__global__ void tridiagonal_periodic(
    const double *d_xl, const double *d_xd, const double *d_xh,
    double *d_c_star, double *d_q_sparse, 
    const double *d_d, double *d_x,
    const int *d_offset, const int REPEAT,
    const int INTERVAL, const int M)
{
    // xl: a
    // xd: b
    // xh: c
    // gamma = 1.0

    double temp, value;

    int n = blockIdx.x * blockDim.x + threadIdx.x;
    while (n < REPEAT)
    {
        const double *_d_d = &d_d[d_offset[n]];
        double       *_d_x = &d_x[d_offset[n]];
        double  *_d_c_star = &d_c_star[d_offset[n]];
        double *_d_q_sparse = &d_q_sparse[d_offset[n]];

        // Forward sweep
        temp = d_xd[0] - 1.0 ;
        _d_c_star[0] = d_xh[0]/temp;
        _d_x[0] = _d_d[0]/temp;
        _d_q_sparse[0] =  1.0/temp;

        for(int i=1; i<M-1; i++)
        {
            _d_c_star[(i-1)*INTERVAL] = d_xh[i-1]/temp;
            temp = d_xd[i]-d_xl[i]*_d_c_star[(i-1)*INTERVAL];
            _d_x[i*INTERVAL] = (_d_d[i*INTERVAL]-d_xl[i]*_d_x[(i-1)*INTERVAL])/temp;
            _d_q_sparse[i*INTERVAL] =   (-d_xl[i]*_d_q_sparse[(i-1)*INTERVAL])/temp;
        }

        _d_c_star[(M-2)*INTERVAL] = d_xh[M-2]/temp;
        temp = d_xd[M-1]-d_xh[M-1]*d_xl[0] - d_xl[M-1]*_d_c_star[(M-2)*INTERVAL];
        _d_x[(M-1)*INTERVAL] =    (_d_d[(M-1)*INTERVAL]-d_xl[M-1]*_d_x[(M-2)*INTERVAL])/temp;
        _d_q_sparse[(M-1)*INTERVAL] = (d_xh[M-1]-d_xl[M-1]*_d_q_sparse[(M-2)*INTERVAL])/temp;

        // Backward substitution
        for(int i=M-2;i>=0; i--)
        {
            _d_x[i*INTERVAL] = _d_x[i*INTERVAL] - _d_c_star[i*INTERVAL]*_d_x[(i+1)*INTERVAL];
            _d_q_sparse[i*INTERVAL] = _d_q_sparse[i*INTERVAL] - _d_c_star[i*INTERVAL]*_d_q_sparse[(i+1)*INTERVAL];
        }

        value = (_d_x[0]+d_xl[0]*_d_x[(M-1)*INTERVAL])/(1.0+_d_q_sparse[0]+d_xl[0]*_d_q_sparse[(M-1)*INTERVAL]);
        for(int i=0; i<M; i++)
            _d_x[i*INTERVAL] = _d_x[i*INTERVAL] - _d_q_sparse[i*INTERVAL]*value;

        n += blockDim.x * gridDim.x;
    }
}