/*-------------------------------------------------------------
* This class defines simulation grids and Lengths parameters and provide
* methods that compute inner product in a given geometry.
*--------------------------------------------------------------*/
#include <iostream>
#include <complex>
#include <hip/hip_runtime.h>
#include <cub/device/device_reduce.cuh>
#include <hip/hip_complex.h>

#include "CudaComputationBox.h"
#include "CudaCommon.h"

//----------------- Constructor -----------------------------
template <typename T>
CudaComputationBox<T>::CudaComputationBox(
    std::vector<int> nx, std::vector<double> lx, std::vector<std::string> bc, const double* mask)
    : ComputationBox(nx, lx, bc, mask)
{
    initialize();
}
template <typename T>
void CudaComputationBox<T>::initialize()
{
    gpu_error_check(hipMalloc((void**)&d_dv, sizeof(double)*this->total_grid));
    gpu_error_check(hipMemcpy(d_dv, dv, sizeof(double)*this->total_grid, hipMemcpyHostToDevice));

    // Temporal storage
    gpu_error_check(hipMalloc((void**)&d_multiple, sizeof(T)*this->total_grid));

    // Allocate memory for cub reduction sum
    gpu_error_check(hipMalloc((void**)&d_sum,      sizeof(T)*this->total_grid));
    gpu_error_check(hipMalloc((void**)&d_sum_out,  sizeof(T)));

    // Determine temporary storage size for cub reduction
    if constexpr (std::is_same<T, double>::value) 
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, this->total_grid);
    else
        hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, this->total_grid, ComplexSumOp(), CuDeviceData<T>{0.0,0.0});
    gpu_error_check(hipMalloc(&d_temp_storage, temp_storage_bytes));
}
//----------------- Destructor -----------------------------
template <typename T>
CudaComputationBox<T>::~CudaComputationBox()
{
    hipFree(d_dv);
    hipFree(d_multiple);
    hipFree(d_sum);
    hipFree(d_sum_out);
    hipFree(d_temp_storage);
}
//-----------------------------------------------------------
template <typename T>
void CudaComputationBox<T>::set_lx(std::vector<double> new_lx)
{
    ComputationBox::set_lx(new_lx);
    gpu_error_check(hipMemcpy(d_dv, this->dv, sizeof(double)*this->total_grid, hipMemcpyHostToDevice));
}
//-----------------------------------------------------------
template <typename T>
T CudaComputationBox<T>::integral_device(const CuDeviceData<T> *d_g)
{
    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    T sum{0.0};

    ker_multi<<<N_BLOCKS, N_THREADS>>>(d_sum, d_g, d_dv, 1.0, this->total_grid);

    if constexpr (std::is_same<T, double>::value)
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, this->total_grid);
    else
        hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, this->total_grid, ComplexSumOp(), CuDeviceData<T>{0.0,0.0});
    gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(T), hipMemcpyDeviceToHost));
    return sum;
}
//-----------------------------------------------------------
template <typename T>
T CudaComputationBox<T>::inner_product_device(const CuDeviceData<T>* d_g, const CuDeviceData<T>* d_h)
{
    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    T sum{0.0};

    ker_multi<<<N_BLOCKS, N_THREADS>>>(d_sum, d_g, d_h, 1.0, this->total_grid);
    ker_multi<<<N_BLOCKS, N_THREADS>>>(d_sum, d_sum, d_dv, 1.0, this->total_grid);

    if constexpr (std::is_same<T, double>::value)
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, this->total_grid);
    else
        hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, this->total_grid, ComplexSumOp(), CuDeviceData<T>{0.0,0.0});

    gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(T), hipMemcpyDeviceToHost));
    return sum;
}
//-----------------------------------------------------------
template <typename T>
T CudaComputationBox<T>::inner_product_inverse_weight_device(const CuDeviceData<T>* d_g, const CuDeviceData<T>* d_h, const CuDeviceData<T>* d_w)
{
    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    T sum{0.0};

    ker_multi<<<N_BLOCKS, N_THREADS>>>(d_sum, d_g, d_h, 1.0, this->total_grid);
    ker_multi<<<N_BLOCKS, N_THREADS>>>(d_sum, d_sum, d_dv, 1.0, this->total_grid);
    ker_divide<<<N_BLOCKS, N_THREADS>>>(d_sum, d_sum, d_w, 1.0, this->total_grid);
    if constexpr (std::is_same<T, double>::value)
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, this->total_grid);
    else
        hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, this->total_grid, ComplexSumOp(), CuDeviceData<T>{0.0,0.0});

    gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(T), hipMemcpyDeviceToHost));
    return sum;
}
//-----------------------------------------------------------
template <typename T>
T CudaComputationBox<T>::multi_inner_product_device(int n_comp, const CuDeviceData<T>* d_g, const CuDeviceData<T>* d_h)
{
    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    T sum{0.0};

    ker_mutiple_multi<<<N_BLOCKS, N_THREADS>>>(n_comp, d_sum, d_g, d_h, 1.0, this->total_grid);
    ker_multi<<<N_BLOCKS, N_THREADS>>>(d_sum, d_sum, d_dv, 1.0, this->total_grid);
    if constexpr (std::is_same<T, double>::value)
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, this->total_grid);
    else
        hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, this->total_grid, ComplexSumOp(), CuDeviceData<T>{0.0,0.0});
    gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(T), hipMemcpyDeviceToHost));
    return sum;
}
//-----------------------------------------------------------
template <typename T>
void CudaComputationBox<T>::zero_mean_device(CuDeviceData<T>* d_g)
{
    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    CuDeviceData<T> sum{0.0};

    ker_multi<<<N_BLOCKS, N_THREADS>>>(d_sum, d_g, d_dv, 1.0, this->total_grid);

    if constexpr (std::is_same<T, double>::value)
    {
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, this->total_grid);
        gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(T), hipMemcpyDeviceToHost));
    }
    else
    {
        hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, this->total_grid, ComplexSumOp(), CuDeviceData<T>{0.0,0.0});
        gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
        sum.x = -sum.x/this->volume;
        sum.y = -sum.y/this->volume;
    }
    ker_linear_scaling<<<N_BLOCKS, N_THREADS>>>(d_g, d_g, 1.0, sum, this->total_grid);
}

// Explicit template instantiation

template class CudaComputationBox<double>;
template class CudaComputationBox<std::complex<double>>;