/*-------------------------------------------------------------
* This class defines simulation grids and Lengths parameters and provide
* methods that compute inner product in a given geometry.
*--------------------------------------------------------------*/
#include <iostream>
#include <thrust/reduce.h>
#include "CudaComputationBox.h"
#include "CudaCommon.h"

//----------------- Constructor -----------------------------
CudaComputationBox::CudaComputationBox(
    std::vector<int> nx, std::vector<double> lx, std::vector<std::string> bc, const double* mask)
    : ComputationBox(nx, lx, bc, mask)
{
    initialize();
}
void CudaComputationBox::initialize()
{
    gpu_error_check(hipMalloc((void**)&d_dv, sizeof(double)*total_grid));
    gpu_error_check(hipMemcpy(d_dv, dv,      sizeof(double)*total_grid, hipMemcpyHostToDevice));

    // Temporal storage
    gpu_error_check(hipMalloc((void**)&d_multiple, sizeof(double)*total_grid));

    gpu_error_check(hipMalloc((void**)&d_g, sizeof(double)*total_grid));
    gpu_error_check(hipMalloc((void**)&d_h, sizeof(double)*total_grid));
    gpu_error_check(hipMalloc((void**)&d_w, sizeof(double)*total_grid));

    // Allocate memory for cub reduction sum
    gpu_error_check(hipMalloc((void**)&d_sum, sizeof(double)*total_grid));
    gpu_error_check(hipMalloc((void**)&d_sum_out, sizeof(double)));
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, total_grid);
    gpu_error_check(hipMalloc(&d_temp_storage, temp_storage_bytes));
}
//----------------- Destructor -----------------------------
CudaComputationBox::~CudaComputationBox()
{
    hipFree(d_dv);

    hipFree(d_multiple);
    hipFree(d_g);
    hipFree(d_h);
    hipFree(d_w);

    hipFree(d_sum);
    hipFree(d_sum_out);
    hipFree(d_temp_storage);
}
//-----------------------------------------------------------
void CudaComputationBox::set_lx(std::vector<double> new_lx)
{
    ComputationBox::set_lx(new_lx);
    gpu_error_check(hipMemcpy(d_dv, dv,  sizeof(double)*total_grid,hipMemcpyHostToDevice));
}
//-----------------------------------------------------------
double CudaComputationBox::integral_device(const double *d_g)
{
    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    double sum{0};

    multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_g, d_dv, 1.0, total_grid);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, total_grid);
    gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(double),hipMemcpyDeviceToHost));
    return sum;
}
//-----------------------------------------------------------
double CudaComputationBox::inner_product_device(const double* d_g, const double* d_h)
{
    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    double sum{0.0};

    multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_g, d_h, 1.0, total_grid);
    multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_dv, d_sum, 1.0, total_grid);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, total_grid);
    gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(double), hipMemcpyDeviceToHost));
    return sum;
}
//-----------------------------------------------------------
double CudaComputationBox::inner_product_inverse_weight_device(const double* d_g, const double* d_h, const double* d_w)
{
    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    double sum{0.0};

    multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_g, d_h, 1.0, total_grid);
    multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_dv, d_sum, 1.0, total_grid);
    divide_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_sum, d_w, 1.0, total_grid);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, total_grid);
    gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(double), hipMemcpyDeviceToHost));
    return sum;
}
//-----------------------------------------------------------
double CudaComputationBox::multi_inner_product_device(int n_comp, const double* d_g, const double* d_h)
{
    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    double sum{0.0};

    mutiple_multi_real<<<N_BLOCKS, N_THREADS>>>(n_comp, d_sum, d_g, d_h, 1.0, total_grid);
    multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_dv, d_sum, 1.0, total_grid);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, total_grid);
    gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(double), hipMemcpyDeviceToHost));
    return sum;
}
//-----------------------------------------------------------
void CudaComputationBox::zero_mean_device(double* d_g)
{
    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    double sum{0.0};

    multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_dv, d_g, 1.0, total_grid);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, total_grid);
    gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(double), hipMemcpyDeviceToHost));
    linear_scaling_real<<<N_BLOCKS, N_THREADS>>>(d_g, d_g, 1.0, -sum/volume, total_grid);
}