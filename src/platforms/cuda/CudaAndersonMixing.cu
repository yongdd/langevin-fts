#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <thrust/reduce.h>
#include "CudaCommon.h"
#include "CudaComputationBox.h"
#include "CudaCircularBuffer.h"
#include "CudaAndersonMixing.h"

template <typename T>
CudaAndersonMixing<T>::CudaAndersonMixing(
    int n_var, int max_hist, double start_error,
    double mix_min,   double mix_init)
    :AndersonMixing<T>(n_var, max_hist, start_error, mix_min, mix_init)
{
    try
    {
        // const int N_GPUS = CudaCommon::get_instance().get_n_gpus();
        // gpu_error_check(hipSetDevice(0));

        // Number of anderson mixing steps, increases from 0 to this->max_hist
        this->n_anderson = -1;
        // Record history of w in GPU device memory
        d_cb_w_hist = new CudaCircularBuffer<T>(this->max_hist+1, this->n_var);
        // Record history of w_deriv in GPU device memory
        d_cb_w_deriv_hist = new CudaCircularBuffer<T>(this->max_hist+1, this->n_var);
        // Record history of inner_product product of w_deriv in CPU host memory
        cb_w_deriv_dots = new CircularBuffer<T>(this->max_hist+1, this->max_hist+1);

        // define arrays for anderson mixing
        this->u_nm = new T*[this->max_hist];
        for(int i=0; i<this->max_hist; i++)
            this->u_nm[i] = new T[this->max_hist];
        this->v_n = new T[this->max_hist];
        this->a_n = new T[this->max_hist];
        this->w_deriv_dots = new T[this->max_hist+1];

        // Create streams
        gpu_error_check(hipStreamCreate(&streams[0])); // for kernel execution
        gpu_error_check(hipStreamCreate(&streams[1])); // for memcpy

        // Fields arrays
        gpu_error_check(hipMalloc((void**)&d_w_current, sizeof(CuDeviceData<T>)*this->n_var));
        gpu_error_check(hipMalloc((void**)&d_w_new,   sizeof(CuDeviceData<T>)*this->n_var));
        gpu_error_check(hipMalloc((void**)&d_w_deriv, sizeof(CuDeviceData<T>)*this->n_var));
        gpu_error_check(hipMalloc((void**)&d_sum,     sizeof(CuDeviceData<T>)*this->n_var));
        gpu_error_check(hipMalloc((void**)&d_sum_out, sizeof(CuDeviceData<T>)));

        // Allocate memory for cub reduction sum
        d_temp_storage = nullptr; 
        temp_storage_bytes = 0;
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, this->n_var, streams[0]);
        gpu_error_check(hipMalloc(&d_temp_storage, temp_storage_bytes));

        // Reset_count
        reset_count();
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
template <typename T>
CudaAndersonMixing<T>::~CudaAndersonMixing()
{
    delete d_cb_w_hist;
    delete d_cb_w_deriv_hist;
    delete cb_w_deriv_dots;

    for (int i=0; i<this->max_hist; i++)
        delete[] u_nm[i];
    delete[] u_nm;
    delete[] v_n;
    delete[] a_n;
    delete[] w_deriv_dots;

    hipFree(d_w_current);
    hipFree(d_w_deriv);
    hipFree(d_w_new);
    hipFree(d_sum);
    hipFree(d_sum_out);
    hipFree(d_temp_storage);

    // Destroy streams
    hipStreamDestroy(streams[0]);
    hipStreamDestroy(streams[1]);
}
template <typename T>
void CudaAndersonMixing<T>::reset_count()
{
    try
    {
        /* initialize mixing parameter */
        this->mix = this->mix_init;
        /* number of anderson mixing steps, increases from 0 to this->max_hist */
        this->n_anderson = -1;

        d_cb_w_hist->reset();
        d_cb_w_deriv_hist->reset();
        cb_w_deriv_dots->reset();
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
template <typename T>
void CudaAndersonMixing<T>::calculate_new_fields(
    T *w_new,
    T *w_current,
    T *w_deriv,
    double old_error_level,
    double error_level)
{
    try
    {
        const int N_BLOCKS = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();
        
        CuDeviceData<T> *d_w_hist1;
        CuDeviceData<T> *d_w_hist2;
        CuDeviceData<T> *d_w_deriv_hist1;
        CuDeviceData<T> *d_w_deriv_hist2;

        gpu_error_check(hipMemcpy(d_w_deriv, w_deriv,  sizeof(T)*this->n_var, hipMemcpyHostToDevice));
        gpu_error_check(hipMemcpy(d_w_current, w_current, sizeof(T)*this->n_var, hipMemcpyHostToDevice));
        // If (N_GPUS > 1)
        //     gpu_error_check(hipMemcpy(d_w_deriv[1], d_w_deriv[0], sizeof(double)*this->n_var, hipMemcpyDeviceToDevice));

        //printf("this->mix: %f\n", this->mix);
        // Condition to start anderson mixing
        if(error_level < this->start_error || this->n_anderson >= 0)
            this->n_anderson = this->n_anderson + 1;
        if( this->n_anderson >= 0 )
        {
            // Number of histories to use for anderson mixing
            this->n_anderson = std::min(this->max_hist, this->n_anderson);
            
            // store the input and output field (the memory is used in a periodic way)
            d_cb_w_hist->insert(d_w_current);
            d_cb_w_deriv_hist->insert(d_w_deriv);

            // Evaluate w_deriv inner_product products for calculating Unm and Vn in Thompson's paper
            for(int i=0; i<=this->n_anderson; i++)
            {
                ker_multi<<<N_BLOCKS, N_THREADS>>>(d_sum, d_w_deriv, d_cb_w_deriv_hist->get_array(i), 1.0, this->n_var);
                hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, this->n_var);
                gpu_error_check(hipMemcpy(&w_deriv_dots[i], d_sum_out, sizeof(T),hipMemcpyDeviceToHost));
            }

            //print_array(this->max_hist+1, w_deriv_dots);
            cb_w_deriv_dots->insert(w_deriv_dots);
        }

        // Conditions to apply the simple mixing method
        if( this->n_anderson <= 0 )
        {
            // dynamically change mixing parameter
            if (old_error_level < error_level)
                this->mix = std::max(this->mix*0.7, this->mix_min);
            else
                this->mix = this->mix*1.01;

            // Make a simple mixing of input and output fields for the next iteration
            ker_lin_comb<<<N_BLOCKS, N_THREADS>>>(d_w_new, 1.0, d_w_current, this->mix, d_w_deriv, this->n_var);
            gpu_error_check(hipMemcpy(w_new, d_w_new, sizeof(T)*this->n_var, hipMemcpyDeviceToHost));
        }
        else
        {
            // Calculate Unm and Vn
            for(int i=0; i<this->n_anderson; i++)
            {
                v_n[i] = cb_w_deriv_dots->get(0, 0)
                        - cb_w_deriv_dots->get(0, i+1);

                for(int j=0; j<this->n_anderson; j++)
                {
                    u_nm[i][j] = cb_w_deriv_dots->get(0, 0)
                                - cb_w_deriv_dots->get(0, i+1)
                                - cb_w_deriv_dots->get(0, j+1)
                                + cb_w_deriv_dots->get(std::min(i+1, j+1),
                                                    std::abs(i-j));
                }
            }
            //print_array(this->max_hist, v_n);
            //exit(-1);
            this->find_an(u_nm, v_n, a_n, this->n_anderson);

            // Calculate the new field
            d_w_hist1 = d_cb_w_hist->get_array(0);
            d_w_deriv_hist1 = d_cb_w_deriv_hist->get_array(0);
            gpu_error_check(hipMemcpy(d_w_new, d_w_hist1, sizeof(CuDeviceData<T>)*this->n_var,hipMemcpyDeviceToDevice));
            ker_lin_comb<<<N_BLOCKS, N_THREADS>>>(d_w_new, 1.0, d_w_hist1, 1.0, d_w_deriv_hist1, this->n_var);
            for(int i=0; i<this->n_anderson; i++)
            {
                d_w_hist2 = d_cb_w_hist->get_array(i+1);
                d_w_deriv_hist2 = d_cb_w_deriv_hist->get_array(i+1);
                ker_add_lin_comb<<<N_BLOCKS, N_THREADS>>>(d_w_new, a_n[i], d_w_hist2,       -a_n[i], d_w_hist1,       this->n_var);
                ker_add_lin_comb<<<N_BLOCKS, N_THREADS>>>(d_w_new, a_n[i], d_w_deriv_hist2, -a_n[i], d_w_deriv_hist1, this->n_var);
            }
            gpu_error_check(hipMemcpy(w_new, d_w_new, sizeof(T)*this->n_var,hipMemcpyDeviceToHost));
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
template <typename T>
void CudaAndersonMixing<T>::print_array(int n, T *a)
{
    for(int i=0; i<n-1; i++)
    {
        std::cout << a[i] << ", ";
    }
    std::cout << a[n-1] << std::endl;
}

// Explicit template instantiation
template class CudaAndersonMixing<double>;
// template class CudaCircularBuffer<std::complex<double>>;