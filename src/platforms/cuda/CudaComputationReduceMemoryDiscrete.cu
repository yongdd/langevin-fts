#include "hip/hip_runtime.h"
#include <complex>
#include <iostream>
#include <chrono>
#include <omp.h>

#include "CudaComputationBox.h"
#include "CudaComputationReduceMemoryDiscrete.h"
#include "CudaSolverPseudoDiscrete.h"

CudaComputationReduceMemoryDiscrete::CudaComputationReduceMemoryDiscrete(
    ComputationBox<double>* cb,
    Molecules *molecules,
    PropagatorComputationOptimizer *propagator_computation_optimizer)
    : PropagatorComputation(cb, molecules, propagator_computation_optimizer)
{
    try
    {
        #ifndef NDEBUG
        std::cout << "--------- Discrete Chain Solver, GPU Memoery Saving Version ---------" << std::endl;
        #endif

        const int M = this->cb->get_total_grid();
        const int N_GPUS = CudaCommon::get_instance().get_n_gpus();

        // The number of parallel streams for propagator computation
        const char *ENV_OMP_NUM_THREADS = getenv("OMP_NUM_THREADS");
        std::string env_omp_num_threads(ENV_OMP_NUM_THREADS ? ENV_OMP_NUM_THREADS  : "");
        if (env_omp_num_threads.empty())
            n_streams = MAX_STREAMS;
        else
            n_streams =  std::min(std::stoi(env_omp_num_threads), MAX_STREAMS);
        #ifndef NDEBUG
        std::cout << "The number of CPU threads: " << n_streams << std::endl;
        #endif

        // Copy streams
        for(int i=0; i<n_streams; i++)
        {
            gpu_error_check(hipSetDevice(i % N_GPUS));
            gpu_error_check(hipStreamCreate(&streams[i][0])); // for kernel execution
            gpu_error_check(hipStreamCreate(&streams[i][1])); // for memcpy
        }

        this->propagator_solver = new CudaSolverPseudoDiscrete(cb, molecules, n_streams, streams, true);

        // Allocate memory for propagators
        if( this->propagator_computation_optimizer->get_computation_propagators().size() == 0)
            throw_with_line_number("There is no propagator code. Add polymers first.");
        for(const auto& item: this->propagator_computation_optimizer->get_computation_propagators())
        {
             // There are N segments
             // Example (N==5)
             // O--O--O--O--O
             // 1  2  3  4  5

             // Legend)
             // -- : full bond
             // O  : full segment

            std::string key = item.first;
            int max_n_segment = item.second.max_n_segment+1; 
            propagator_size[key] = max_n_segment;

            // Allocate memory for q(r,1/2)
            propagator_half_steps[key] = new double*[max_n_segment];
            propagator_half_steps[key][0] = nullptr;
            if (item.second.deps.size() > 0)
                gpu_error_check(hipHostMalloc((void**)&propagator_half_steps[key][0], sizeof(double)*M));

            // Allocate memory for q(r,s+1/2)
            for(int i=1; i<propagator_size[key]; i++)
            {
                propagator_half_steps[key][i] = nullptr;
                if (item.second.junction_ends.find(i) != item.second.junction_ends.end())
                    gpu_error_check(hipHostMalloc((void**)&propagator_half_steps[key][i], sizeof(double)*M));
            }

            // Allocate memory for q(r,s)
            // Index 0 will be not used
            propagator[key] = new double*[max_n_segment];
            propagator[key][0] = nullptr;
            for(int i=1; i<propagator_size[key]; i++)
                gpu_error_check(hipHostMalloc((void**)&propagator[key][i], sizeof(double)*M));

            #ifndef NDEBUG
            propagator_finished[key] = new bool[max_n_segment];
            for(int i=0; i<max_n_segment;i++)
                propagator_finished[key][i] = false;
            for (int n: item.second.junction_ends)
                propagator_half_steps_finished[key][n] = false;
            #endif
        }

        // Allocate memory for concentrations
        if( this->propagator_computation_optimizer->get_computation_blocks().size() == 0)
            throw_with_line_number("There is no block. Add polymers first.");
        for(const auto& item: this->propagator_computation_optimizer->get_computation_blocks())
        {
            phi_block[item.first] = nullptr;
            gpu_error_check(hipHostMalloc((void**)&phi_block[item.first], sizeof(double)*M));
        }

        // Remember one segment for each polymer chain to compute total partition function
        int current_p = 0;
        for(const auto& block: phi_block)
        {
            const auto& key = block.first;
            int p                 = std::get<0>(key);
            std::string key_left  = std::get<1>(key);
            std::string key_right = std::get<2>(key);

            // Skip if already found one segment
            if (p != current_p)
                continue;

            int n_aggregated = this->propagator_computation_optimizer->get_computation_block(key).v_u.size()/
                               this->propagator_computation_optimizer->get_computation_block(key).n_repeated;
            int n_segment_left = this->propagator_computation_optimizer->get_computation_block(key).n_segment_left;
            std::string monomer_type = this->propagator_computation_optimizer->get_computation_block(key).monomer_type;

            // Skip if n_segment_left is 0
            if (n_segment_left == 0)
                continue;

            single_partition_segment.push_back(std::make_tuple(
                p,
                propagator[key_left][n_segment_left],    // q
                propagator[key_right][1],                  // q_dagger
                monomer_type,       
                n_aggregated                               // how many propagators are aggregated
                ));
            current_p++;
        }

       // Find propagators and bond length for each segment to prepare stress computation
        for(const auto& block: phi_block)
        {
            const auto& key = block.first;
            int p                 = std::get<0>(key);
            std::string key_left  = std::get<1>(key);
            std::string key_right = std::get<2>(key);

            const int N_RIGHT = this->propagator_computation_optimizer->get_computation_block(key).n_segment_right;
            const int N_LEFT  = this->propagator_computation_optimizer->get_computation_block(key).n_segment_left;

            // If there is no segment
            if(N_RIGHT == 0)
                continue;

            double **q_1 = propagator[key_left];     // dependency v
            double **q_2 = propagator[key_right];    // dependency u

            auto& _block_stress_compuation_key = block_stress_computation_plan[key];

            // Find propagators and bond length
            for(int n=0; n<=N_RIGHT; n++)
            {
                double *propagator_left  = nullptr;
                double *propagator_right = nullptr;
                bool is_half_bond_length = false;

                // At v
                if (n == N_LEFT)
                {
                    if (this->propagator_computation_optimizer->get_computation_propagator(key_left).deps.size() == 0) // if v is leaf node, skip
                    {
                        _block_stress_compuation_key.push_back(std::make_tuple(propagator_left, propagator_right, is_half_bond_length));
                        continue;
                    }
                    
                    propagator_left  = propagator_half_steps[key_left][0];
                    propagator_right = q_2[N_RIGHT];
                    is_half_bond_length = true;
                }
                // At u
                else if (n == 0 && N_LEFT == N_RIGHT){
                    if (this->propagator_computation_optimizer->get_computation_propagator(key_right).deps.size() == 0) // if u is leaf node, skip
                    {
                        _block_stress_compuation_key.push_back(std::make_tuple(propagator_left, propagator_right, is_half_bond_length));
                        continue;
                    }

                    propagator_left  = q_1[N_LEFT];
                    propagator_right = propagator_half_steps[key_right][0];
                    is_half_bond_length = true;
                }
                // At aggregation junction
                else if (n == 0)
                {
                    _block_stress_compuation_key.push_back(std::make_tuple(propagator_left, propagator_right, is_half_bond_length));
                    continue;
                }
                // Within the blocks
                else
                {
                    propagator_left  = q_1[N_LEFT-n];
                    propagator_right = q_2[n];
                    is_half_bond_length = false;
                }
                _block_stress_compuation_key.push_back(std::make_tuple(propagator_left, propagator_right, is_half_bond_length));
            }
        }

        // Concentrations for each solvent
        for(int s=0;s<this->molecules->get_n_solvent_types();s++)
            phi_solvent.push_back(new double[M]);

        // Create scheduler for computation of propagator
        sc = new Scheduler(this->propagator_computation_optimizer->get_computation_propagators(), n_streams); 

        // Allocate memory for pseudo-spectral: advance_propagator()
        double q_unity[M];
        for(int i=0; i<M; i++)
            q_unity[i] = 1.0;
        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            gpu_error_check(hipMalloc((void**)&d_q_unity[gpu], sizeof(double)*M));
            gpu_error_check(hipMemcpy(d_q_unity[gpu], q_unity, sizeof(double)*M, hipMemcpyHostToDevice));
        }

        // Allocate memory for propagator computation
        for(int i=0; i<n_streams; i++)
        {
            gpu_error_check(hipSetDevice(i % N_GPUS));
            gpu_error_check(hipMalloc((void**)&d_q_one[i][0], sizeof(double)*M)); // for prev
            gpu_error_check(hipMalloc((void**)&d_q_one[i][1], sizeof(double)*M)); // for next
            gpu_error_check(hipMalloc((void**)&d_propagator_sub_dep[i][0], sizeof(double)*M)); // for prev
            gpu_error_check(hipMalloc((void**)&d_propagator_sub_dep[i][1], sizeof(double)*M)); // for next
        }

        gpu_error_check(hipSetDevice(0));
        // For concentration computation
        gpu_error_check(hipMalloc((void**)&d_q_block_v[0], sizeof(double)*M)); // for prev
        gpu_error_check(hipMalloc((void**)&d_q_block_v[1], sizeof(double)*M)); // for next
        gpu_error_check(hipMalloc((void**)&d_q_block_u[0], sizeof(double)*M)); // for prev
        gpu_error_check(hipMalloc((void**)&d_q_block_u[1], sizeof(double)*M)); // for next
        gpu_error_check(hipMalloc((void**)&d_phi,          sizeof(double)*M));

        // Allocate memory for stress calculation: compute_stress()
        for(int i=0; i<n_streams; i++)
        {
            gpu_error_check(hipSetDevice(i % N_GPUS));
            gpu_error_check(hipMalloc((void**)&d_q_pair[i][0], sizeof(double)*2*M)); // prev
            gpu_error_check(hipMalloc((void**)&d_q_pair[i][1], sizeof(double)*2*M)); // next
        }

        // Copy mask to d_q_mask
        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            if (this->cb->get_mask() != nullptr)
            {
                gpu_error_check(hipMalloc((void**)&d_q_mask [gpu], sizeof(double)*M));
                gpu_error_check(hipMemcpy(d_q_mask[gpu], this->cb->get_mask(), sizeof(double)*M, hipMemcpyHostToDevice));
            }
            else
                d_q_mask[gpu] = nullptr;
        }

        propagator_solver->update_laplacian_operator();
        gpu_error_check(hipSetDevice(0));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
CudaComputationReduceMemoryDiscrete::~CudaComputationReduceMemoryDiscrete()
{
    const int N_GPUS = CudaCommon::get_instance().get_n_gpus();

    delete propagator_solver;
    delete sc;

    for(const auto& item: propagator)
    {
        for(int i=0; i<propagator_size[item.first]; i++)
        {
            if(item.second[i] != nullptr)
                hipHostFree(item.second[i]);
        }
        delete[] item.second;
    }
    for(const auto& item: propagator_half_steps)
    {
        for(int i=0; i<propagator_size[item.first]; i++)
        {
            if(item.second[i] != nullptr)
                hipHostFree(item.second[i]);
        }
        delete[] item.second;
    }

    for(const auto& item: phi_block)
        hipHostFree(item.second);
    for(const auto& item: phi_solvent)
        delete[] item;

    #ifndef NDEBUG
    for(const auto& item: propagator_finished)
        delete[] item.second;
    #endif

    // For pseudo-spectral: advance_one_propagator()
    for(int i=0; i<n_streams; i++)
    {
        hipFree(d_q_one[i][0]); // for prev
        hipFree(d_q_one[i][1]); // for next
        hipFree(d_propagator_sub_dep[i][0]); // for prev
        hipFree(d_propagator_sub_dep[i][1]); // for next
    }

    // For stress calculation: compute_stress()
    for(int i=0; i<n_streams; i++)
    {
        hipFree(d_q_pair[i][0]);
        hipFree(d_q_pair[i][1]);
    }

    // For concentration computation
    hipFree(d_q_block_v[0]);
    hipFree(d_q_block_v[1]);
    hipFree(d_q_block_u[0]);
    hipFree(d_q_block_u[1]);
    hipFree(d_phi);

    // For pseudo-spectral: advance_propagator()
    for(int gpu=0; gpu<N_GPUS; gpu++)
    {
        if (d_q_mask[gpu] != nullptr)
            hipFree(d_q_mask[gpu]);
        hipFree(d_q_unity[gpu]);
    }
    
    // Destroy streams
    for(int i=0; i<n_streams; i++)
    {
        hipStreamDestroy(streams[i][0]);
        hipStreamDestroy(streams[i][1]);
    }
}

void CudaComputationReduceMemoryDiscrete::update_laplacian_operator()
{
    try
    {
        propagator_solver->update_laplacian_operator();
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}

void CudaComputationReduceMemoryDiscrete::compute_statistics(
    std::map<std::string, const double*> w_input,
    std::map<std::string, const double*> q_init)
{
    this->compute_propagators(w_input, q_init);
    this->compute_concentrations();
}

void CudaComputationReduceMemoryDiscrete::compute_propagators(
    std::map<std::string, const double*> w_input,
    std::map<std::string, const double*> q_init)
{
    try{
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();
        const int N_GPUS = CudaCommon::get_instance().get_n_gpus();

        const int M = this->cb->get_total_grid();
        const double ds = this->molecules->get_ds();

        std::string device = "cpu";
        hipMemcpyKind cudaMemcpyInputToDevice;
        if (device == "gpu")
            cudaMemcpyInputToDevice = hipMemcpyDeviceToDevice;
        else if(device == "cpu")
            cudaMemcpyInputToDevice = hipMemcpyHostToDevice;
        else
        {
            throw_with_line_number("Invalid device \"" + device + "\".");
        }

        for(const auto& item: this->propagator_computation_optimizer->get_computation_propagators())
        {
            if( w_input.find(item.second.monomer_type) == w_input.end())
                throw_with_line_number("monomer_type \"" + item.second.monomer_type + "\" is not in w_input.");
        }

        // Update dw or d_exp_dw
        propagator_solver->update_dw(device, w_input);

        // For each time span
        auto& branch_schedule = sc->get_schedule();
        for (auto parallel_job = branch_schedule.begin(); parallel_job != branch_schedule.end(); parallel_job++)
        {
            // // display all jobs
            // #ifndef NDEBUG
            // std::cout << "jobs:" << std::endl;
            // for(size_t job=0; job<parallel_job->size(); job++)
            // {
            //     auto& key = std::get<0>((*parallel_job)[job]);
            //     int n_segment_from = std::get<1>((*parallel_job)[job]);
            //     int n_segment_to = std::get<2>((*parallel_job)[job]);
            //     std::cout << "key, n_segment_from, n_segment_to: " + key + ", " + std::to_string(n_segment_from) + ", " + std::to_string(n_segment_to) + ". " << std::endl;
            //     std::cout << "half_steps: ";
            //     std::cout << "{";
            //     for (auto it = propagator_half_steps[key].begin(); it != propagator_half_steps[key].end(); ++it)
            //     {
            //         std::cout << it->first+1;
            //         if (std::next(it) != propagator_half_steps[key].end()) {
            //             std::cout << ", ";
            //         }
            //     }
            //     std::cout << "}, "<< std::endl;
            // }
            // auto start_time = std::chrono::duration_cast<std::chrono::microseconds>
            //     (std::chrono::system_clock::now().time_since_epoch()).count();
            // #endif

            // For each propagator
            #pragma omp parallel for num_threads(n_streams)
            for(size_t job=0; job<parallel_job->size(); job++)
            {
                const int STREAM = omp_get_thread_num();
                int gpu = omp_get_thread_num() % N_GPUS;
                gpu_error_check(hipSetDevice(gpu));

                // printf("gpu, STREAM: %d, %d\n ", gpu, STREAM);

                auto& key = std::get<0>((*parallel_job)[job]);
                int n_segment_from = std::get<1>((*parallel_job)[job]);
                int n_segment_to = std::get<2>((*parallel_job)[job]);
                auto& deps = this->propagator_computation_optimizer->get_computation_propagator(key).deps;
                auto monomer_type = this->propagator_computation_optimizer->get_computation_propagator(key).monomer_type;

                // std::cout << "gpu, STREAM, key, n_segment_from, n_segment_to, monomer_type: "
                // if (STREAM == 0)
                //     std::cout << gpu << ", " << STREAM << ", " << n_segment_from << ", " << n_segment_to << ", " << monomer_type << ", " << key << ", "  << std::endl;    

                // #ifndef NDEBUG
                // #pragma omp critical
                // std::cout << job << " started, stream: " << STREAM << ", " <<
                //     std::chrono::duration_cast<std::chrono::microseconds>
                //     (std::chrono::system_clock::now().time_since_epoch()).count() - start_time << std::endl;
                // #endif

                // Check key
                #ifndef NDEBUG
                if (propagator.find(key) == propagator.end())
                    std::cout<< "Could not find key '" + key + "'. " << std::endl;
                #endif

                double **_propagator = propagator[key];
                double *_d_exp_dw = propagator_solver->d_exp_dw[gpu][monomer_type];

                // Calculate one block end
                if(n_segment_from == 0 && deps.size() == 0) // if it is leaf node
                {
                    // #ifndef NDEBUG
                    // #pragma omp critical
                    // std::cout << job << " init 1, " << 
                    //     std::chrono::duration_cast<std::chrono::microseconds>
                    //     (std::chrono::system_clock::now().time_since_epoch()).count() - start_time << std::endl;
                    // #endif

                    // q_init
                    if (key[0] == '{')
                    {
                        std::string g = PropagatorCode::get_q_input_idx_from_key(key);
                        if (q_init.find(g) == q_init.end())
                            std::cout<<  "Could not find q_init[\"" + g + "\"]." << std::endl;
                        gpu_error_check(hipMemcpy(d_q_one[STREAM][0], q_init[g], sizeof(double)*M, cudaMemcpyInputToDevice));
                        ker_multi<double><<<N_BLOCKS, N_THREADS>>>(d_q_one[STREAM][0], d_q_one[STREAM][0], _d_exp_dw, 1.0, M);
                    }
                    else
                    {
                        gpu_error_check(hipMemcpy(d_q_one[STREAM][0], _d_exp_dw, sizeof(double)*M, hipMemcpyDeviceToDevice));
                    }

                    #ifndef NDEBUG
                    propagator_finished[key][1] = true;
                    #endif
                }
                else if (n_segment_from == 0 && deps.size() > 0) // if it is not leaf node
                {
                    // If it is aggregated
                    if (key[0] == '[')
                    {
                        // #ifndef NDEBUG
                        // #pragma omp critical
                        // std::cout << job << " init 2, " << 
                        //     std::chrono::duration_cast<std::chrono::microseconds>
                        //     (std::chrono::system_clock::now().time_since_epoch()).count() - start_time << std::endl;
                        // #endif

                        // Initialize to zero
                        gpu_error_check(hipMemset(d_q_one[STREAM][0], 0, sizeof(double)*M));

                        int prev, next;
                        prev = 0;
                        next = 1;

                        // Copy memory from host to device
                        std::string sub_dep = std::get<0>(deps[0]);
                        int sub_n_segment   = std::get<1>(deps[0]);
                        int sub_n_repeated;

                        if (sub_n_segment == 0)
                        {
                            gpu_error_check(hipMemcpy(d_propagator_sub_dep[STREAM][prev], propagator_half_steps[sub_dep][0], sizeof(double)*M, hipMemcpyHostToDevice));
                        }
                        else
                        {
                            gpu_error_check(hipMemcpy(d_propagator_sub_dep[STREAM][prev], propagator[sub_dep][sub_n_segment], sizeof(double)*M, hipMemcpyHostToDevice));
                        }

                        for(size_t d=0; d<deps.size(); d++)
                        {
                            sub_dep         = std::get<0>(deps[d]);
                            sub_n_segment   = std::get<1>(deps[d]);
                            sub_n_repeated  = std::get<2>(deps[d]);
                            double **_propagator_sub_dep_next;

                            // STREAM 1: copy memory from host to device
                            if (d < deps.size()-1)
                            {
                                std::string sub_dep_next = std::get<0>(deps[d+1]);
                                int sub_n_segment_next   = std::get<1>(deps[d+1]);

                                if (sub_n_segment == 0)
                                {
                                    // Check sub key
                                    #ifndef NDEBUG
                                    if (propagator_half_steps.find(sub_dep_next) == propagator_half_steps.end())
                                        std::cout << "Could not find sub key '" + sub_dep_next + "'. " << std::endl;
                                    if (!propagator_half_steps_finished[sub_dep_next][0])
                                        std::cout << "Could not compute '" + key +  "', since '"+ sub_dep_next + std::to_string(0) + "' is not prepared." << std::endl;
                                    #endif

                                    _propagator_sub_dep_next = propagator_half_steps[sub_dep_next];
                                }
                                else
                                {
                                    // Check sub key
                                    #ifndef NDEBUG
                                    if (propagator.find(sub_dep_next) == propagator.end())
                                        std::cout<< "Could not find sub key '" + sub_dep_next + "'. " << std::endl;
                                    if (!propagator_finished[sub_dep_next][sub_n_segment_next])
                                        std::cout<< "Could not compute '" + key +  "', since '"+ sub_dep_next + std::to_string(sub_n_segment) + "' is not prepared." << std::endl;
                                    #endif

                                    _propagator_sub_dep_next = propagator[sub_dep_next];
                                }

                                gpu_error_check(hipMemcpyAsync(d_propagator_sub_dep[STREAM][next],
                                                _propagator_sub_dep_next[sub_n_segment_next], sizeof(double)*M,
                                                hipMemcpyHostToDevice, streams[STREAM][1]));
                            }

                            // STREAM 0: compute linear combination
                            ker_lin_comb<double><<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
                                    d_q_one[STREAM][0], 1.0, d_q_one[STREAM][0],
                                    sub_n_repeated, d_propagator_sub_dep[STREAM][prev], M);

                            std::swap(prev, next);
                            hipDeviceSynchronize();
                        }

                        // if sub_n_segment == 0
                        if (std::get<1>(deps[0]) == 0)
                        {
                            gpu_error_check(hipMemcpyAsync(propagator_half_steps[key][0], d_q_one[STREAM][0], sizeof(double)*M, hipMemcpyDeviceToHost, streams[STREAM][0]));

                            // Add half bond, STREAM 0
                            propagator_solver->advance_propagator_half_bond_step(
                                gpu, STREAM,
                                d_q_one[STREAM][0], d_q_one[STREAM][0], monomer_type);

                            // Add full segment
                            ker_multi<double><<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(d_q_one[STREAM][0], d_q_one[STREAM][0], _d_exp_dw, 1.0, M);
                        }
                        else
                        {
                            propagator_solver->advance_propagator(
                                gpu, STREAM,
                                d_q_one[STREAM][0], d_q_one[STREAM][0],
                                monomer_type, d_q_mask[gpu]);
                        }

                        #ifndef NDEBUG
                        propagator_finished[key][1] = true;
                        #endif
                    }
                    else
                    {
                        // #ifndef NDEBUG
                        // #pragma omp critical
                        // std::cout << job << " init 3, " << 
                        //     std::chrono::duration_cast<std::chrono::microseconds>
                        //     (std::chrono::system_clock::now().time_since_epoch()).count() - start_time << std::endl;
                        // #endif

                        // Combine branches
                        // Initialize to one
                        gpu_error_check(hipMemcpy(d_q_one[STREAM][0], d_q_unity[gpu], sizeof(double)*M, hipMemcpyDeviceToDevice));

                        int prev, next;
                        prev = 0;
                        next = 1;

                        // Copy memory from host to device
                        std::string sub_dep = std::get<0>(deps[0]);
                        int sub_n_segment   = std::get<1>(deps[0]);
                        gpu_error_check(hipMemcpy(d_propagator_sub_dep[STREAM][prev], propagator_half_steps[sub_dep][sub_n_segment], sizeof(double)*M, hipMemcpyHostToDevice));

                        for(size_t d=0; d<deps.size(); d++)
                        {
                            sub_dep       = std::get<0>(deps[d]);
                            sub_n_segment = std::get<1>(deps[d]);

                            // Check sub key
                            #ifndef NDEBUG
                            if (!propagator_half_steps_finished[sub_dep][sub_n_segment])
                                std::cout<< "Could not compute '" + key +  "', since '"+ sub_dep + std::to_string(sub_n_segment) + "+1/2' is not prepared." << std::endl;
                            #endif

                            // STREAM 1: copy memory from host to device
                            if (d < deps.size()-1)
                            {
                                std::string sub_dep_next = std::get<0>(deps[d+1]);
                                int sub_n_segment_next   = std::get<1>(deps[d+1]);

                                gpu_error_check(hipMemcpyAsync(d_propagator_sub_dep[STREAM][next],
                                                propagator_half_steps[sub_dep_next][sub_n_segment_next], sizeof(double)*M,
                                                hipMemcpyHostToDevice, streams[STREAM][1]));
                            }

                            // STREAM 0: multiply 
                            ker_multi<double><<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
                                d_q_one[STREAM][0], d_q_one[STREAM][0], d_propagator_sub_dep[STREAM][prev], 1.0, M);

                            std::swap(prev, next);
                            hipDeviceSynchronize();
                        }
                        gpu_error_check(hipMemcpy(propagator_half_steps[key][0], d_q_one[STREAM][0], sizeof(double)*M,hipMemcpyDeviceToHost));

                        #ifndef NDEBUG
                        propagator_half_steps_finished[key][0] = true;
                        #endif

                        if (n_segment_to > 0)
                        {
                            // Add half bond
                            propagator_solver->advance_propagator_half_bond_step(
                                gpu, STREAM,
                                d_q_one[STREAM][0], d_q_one[STREAM][0], monomer_type);

                            // Add full segment
                            ker_multi<double><<<N_BLOCKS, N_THREADS>>>(d_q_one[STREAM][0], d_q_one[STREAM][0], _d_exp_dw, 1.0, M);

                            #ifndef NDEBUG
                            propagator_finished[key][1] = true;
                            #endif
                        }
                    }
                }

                if (n_segment_to == 0)
                {
                    gpu_error_check(hipStreamSynchronize(streams[STREAM][0]));
                    gpu_error_check(hipStreamSynchronize(streams[STREAM][1]));
                    continue;
                }

                if (n_segment_from == 0)
                {
                    // Multiply mask
                    if (d_q_mask[gpu] != nullptr)
                        ker_multi<double><<<N_BLOCKS, N_THREADS>>>(d_q_one[STREAM][0], d_q_one[STREAM][0], d_q_mask[gpu], 1.0, M);

                    // Copy data between device and host
                    gpu_error_check(hipMemcpy(_propagator[1], d_q_one[STREAM][0], sizeof(double)*M, hipMemcpyDeviceToHost));

                    // q(r, 1+1/2)
                    if (propagator_half_steps[key][1] != nullptr)
                    {
                        #ifndef NDEBUG
                        if (propagator_half_steps_finished[key][1])
                            std::cout << "already half_step finished: " + key + ", " + std::to_string(1) << std::endl;
                        #endif

                        propagator_solver->advance_propagator_half_bond_step(
                            gpu, STREAM,
                            d_q_one[STREAM][0],
                            d_q_one[STREAM][1],
                            monomer_type);

                        gpu_error_check(hipMemcpy(
                            propagator_half_steps[key][1],
                            d_q_one[STREAM][1],
                            sizeof(double)*M, hipMemcpyDeviceToHost));

                        #ifndef NDEBUG
                        propagator_half_steps_finished[key][1] = true;
                        #endif
                    }
                    n_segment_from++;
                }
                else
                {
                    // Copy data between device and host
                    gpu_error_check(hipMemcpy(d_q_one[STREAM][0], _propagator[n_segment_from], sizeof(double)*M, hipMemcpyHostToDevice));
                }

                int prev, next;
                prev = 0;
                next = 1;

                // Create events
                hipEvent_t kernel_done;
                hipEvent_t memcpy_done;
                gpu_error_check(hipEventCreate(&kernel_done));
                gpu_error_check(hipEventCreate(&memcpy_done));

                // q(r,s)
                for(int n=n_segment_from; n<n_segment_to; n++)
                {
                    #ifndef NDEBUG
                    if (!propagator_finished[key][n])
                        std::cout << "unfinished, key: " + key + ", " + std::to_string(n) << std::endl;
                    if (propagator_finished[key][n+1])
                        std::cout << "already finished: " + key + ", " + std::to_string(n+1) << std::endl;
                    #endif

                    // #ifndef NDEBUG
                    // #pragma omp critical
                    // std::cout << job << " q_s, " << n << ", " << 
                    //     std::chrono::duration_cast<std::chrono::microseconds>
                    //     (std::chrono::system_clock::now().time_since_epoch()).count() - start_time << std::endl;
                    // #endif

                    // STREAM 0: calculate propagators
                    propagator_solver->advance_propagator(
                        gpu, STREAM, 
                        d_q_one[STREAM][prev],
                        d_q_one[STREAM][next],
                        monomer_type, d_q_mask[gpu]);
                    gpu_error_check(hipEventRecord(kernel_done, streams[STREAM][0]));

                    // STREAM 1: copy memory from device to host
                    if (n > n_segment_from)
                    {
                        gpu_error_check(hipMemcpyAsync(
                           _propagator[n],
                            d_q_one[STREAM][prev],
                            sizeof(double)*M, hipMemcpyDeviceToHost, streams[STREAM][1]));
                        gpu_error_check(hipEventRecord(memcpy_done, streams[STREAM][1]));
                    }

                    // Wait until computation and memory copy are done
                    gpu_error_check(hipStreamWaitEvent(streams[STREAM][1], kernel_done, 0));
                    gpu_error_check(hipStreamWaitEvent(streams[STREAM][0], memcpy_done, 0));

                    std::swap(prev, next);

                    #ifndef NDEBUG
                    propagator_finished[key][n+1] = true;
                    #endif
                }
                // Copy memory from device to host
                gpu_error_check(hipMemcpyAsync(
                    _propagator[n_segment_to],
                    d_q_one[STREAM][prev],
                    sizeof(double)*M, hipMemcpyDeviceToHost, streams[STREAM][1]));

                gpu_error_check(hipEventRecord(memcpy_done, streams[STREAM][1]));
                gpu_error_check(hipStreamWaitEvent(streams[STREAM][0], memcpy_done, 0));

                gpu_error_check(hipEventDestroy(kernel_done));
                gpu_error_check(hipEventDestroy(memcpy_done));

                // q(r, s+1/2)
                for(int n=n_segment_from; n<n_segment_to; n++)
                {
                    if (propagator_half_steps[key][n+1] != nullptr)
                    {
                        // #ifndef NDEBUG
                        // #pragma omp critical
                        // std::cout << job << " q_s+1/2, " << n << ", " << 
                        //     std::chrono::duration_cast<std::chrono::microseconds>
                        //     (std::chrono::system_clock::now().time_since_epoch()).count() - start_time << std::endl;
                        // #endif

                        #ifndef NDEBUG
                        if (propagator_half_steps_finished[key][n+1])
                            std::cout << "already half_step finished: " + key + ", " + std::to_string(n+1) << std::endl;
                        #endif

                        gpu_error_check(hipMemcpyAsync(
                            d_q_one[STREAM][0], _propagator[n+1],
                            sizeof(double)*M, hipMemcpyHostToDevice, streams[STREAM][0]));

                        propagator_solver->advance_propagator_half_bond_step(
                            gpu, STREAM,
                            d_q_one[STREAM][0],
                            d_q_one[STREAM][1],
                            monomer_type);

                        gpu_error_check(hipMemcpyAsync(
                            propagator_half_steps[key][n+1],
                            d_q_one[STREAM][1],
                            sizeof(double)*M, hipMemcpyDeviceToHost, streams[STREAM][0]));

                        #ifndef NDEBUG
                        propagator_half_steps_finished[key][n+1] = true;
                        #endif
                    }
                }
                gpu_error_check(hipStreamSynchronize(streams[STREAM][0]));
                gpu_error_check(hipStreamSynchronize(streams[STREAM][1]));
            }

            // Synchronize all GPUs
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                gpu_error_check(hipDeviceSynchronize());
            }
        }

        gpu_error_check(hipSetDevice(0));
        // Compute total partition function of each distinct polymers
        for(const auto& segment_info: single_partition_segment)
        {
            int p                    = std::get<0>(segment_info);
            double *propagator_left  = std::get<1>(segment_info);
            double *propagator_right = std::get<2>(segment_info);
            std::string monomer_type = std::get<3>(segment_info);
            int n_aggregated         = std::get<4>(segment_info);
            double *_d_exp_dw = propagator_solver->d_exp_dw[0][monomer_type];

            // Copy propagators from host to device
            gpu_error_check(hipMemcpy(d_q_block_v[0], propagator_left,  sizeof(double)*M, hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_q_block_u[0], propagator_right, sizeof(double)*M, hipMemcpyHostToDevice));

            this->single_polymer_partitions[p] = this->cb->inner_product_inverse_weight_device(
                d_q_block_v[0],  // q
                d_q_block_u[0],  // q^dagger
                _d_exp_dw)/n_aggregated/this->cb->get_volume();
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}

void CudaComputationReduceMemoryDiscrete::compute_concentrations()
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();
        const int N_GPUS = CudaCommon::get_instance().get_n_gpus();

        const int M = this->cb->get_total_grid();

        // Calculate segment concentrations
        for(const auto& block: phi_block)
        {
            const auto& key = block.first;
            int p                 = std::get<0>(key);
            std::string key_left  = std::get<1>(key);
            std::string key_right = std::get<2>(key);

            int n_segment_right      = this->propagator_computation_optimizer->get_computation_block(key).n_segment_right;
            int n_segment_left       = this->propagator_computation_optimizer->get_computation_block(key).n_segment_left;
            std::string monomer_type = this->propagator_computation_optimizer->get_computation_block(key).monomer_type;
            int n_repeated           = this->propagator_computation_optimizer->get_computation_block(key).n_repeated;
            double *_d_exp_dw = propagator_solver->d_exp_dw[0][monomer_type];

            // If there is no segment
            if(n_segment_right == 0)
            {
                for(int i=0; i<M;i++)
                    block.second[i] = 0.0;
                continue;
            }

            // Check keys
            #ifndef NDEBUG
            if (propagator.find(key_left) == propagator.end())
                throw_with_line_number("Could not find key_left key'" + key_left + "'. ");
            if (propagator.find(key_right) == propagator.end())
                throw_with_line_number("Could not find key_right key'" + key_right + "'. ");
            #endif

            // Normalize concentration
            Polymer& pc = this->molecules->get_polymer(p);
            double norm = this->molecules->get_ds()*pc.get_volume_fraction()/pc.get_alpha()/this->single_polymer_partitions[p]*n_repeated;

            // Calculate phi of one block (possibly multiple blocks when using aggregation)
            calculate_phi_one_block(
                block.second,               // phi
                propagator[key_left],       // dependency v
                propagator[key_right],      // dependency u
                _d_exp_dw,                  // exp_dw
                n_segment_right,
                n_segment_left,
                norm);
        }
        // Calculate partition functions and concentrations of solvents
        for(int s=0; s<this->molecules->get_n_solvent_types(); s++)
        {
            double volume_fraction   = std::get<0>(this->molecules->get_solvent(s));
            std::string monomer_type = std::get<1>(this->molecules->get_solvent(s));
            double *_d_exp_dw = propagator_solver->d_exp_dw[0][monomer_type];

            this->single_solvent_partitions[s] = this->cb->integral_device(_d_exp_dw)/this->cb->get_volume();
            ker_linear_scaling<double><<<N_BLOCKS, N_THREADS>>>(d_phi, _d_exp_dw, volume_fraction/this->single_solvent_partitions[s], 0.0, M);
            gpu_error_check(hipMemcpy(phi_solvent[s], d_phi, sizeof(double)*M, hipMemcpyDeviceToHost));
        }
        gpu_error_check(hipSetDevice(0));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaComputationReduceMemoryDiscrete::calculate_phi_one_block(
    double *phi, double **q_1, double **q_2, double *d_exp_dw, const int N_RIGHT, const int N_LEFT, const double NORM)
{
    try
    {
        gpu_error_check(hipSetDevice(0));
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();
        const int M = this->cb->get_total_grid();

        int prev, next;
        prev = 0;
        next = 1;

        // Copy propagators from host to device
        gpu_error_check(hipMemcpy(d_q_block_v[prev], q_1[N_LEFT], sizeof(double)*M, hipMemcpyHostToDevice));
        gpu_error_check(hipMemcpy(d_q_block_u[prev], q_2[1],      sizeof(double)*M, hipMemcpyHostToDevice));

        // Initialize to zero
        gpu_error_check(hipMemset(d_phi, 0, sizeof(double)*M));
 
        for(int n=1; n<=N_RIGHT; n++)
        {
            // STREAM 1: copy propagators from host to device
            if (n+1 <=N_RIGHT)
            {
                gpu_error_check(hipMemcpyAsync(d_q_block_v[next], q_1[N_LEFT-(n+1)+1],
                    sizeof(double)*M, hipMemcpyHostToDevice, streams[0][1]));
                gpu_error_check(hipMemcpyAsync(d_q_block_u[next], q_2[(n+1)],
                    sizeof(double)*M, hipMemcpyHostToDevice, streams[0][1]));
            }

            // STREAM 0: multiply two propagators
            ker_add_multi<double><<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(d_phi, d_q_block_v[prev], d_q_block_u[prev], NORM, M);
            std::swap(prev, next);
            hipDeviceSynchronize();
        }
        // divide exp_dw
        ker_divide<double><<<N_BLOCKS, N_THREADS>>>(d_phi, d_phi, d_exp_dw, 1.0, M);

        // Copy propagators from device to host
        gpu_error_check(hipMemcpy(phi, d_phi, sizeof(double)*M, hipMemcpyDeviceToHost));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
double CudaComputationReduceMemoryDiscrete::get_total_partition(int polymer)
{
    try
    {
        return single_polymer_partitions[polymer];
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaComputationReduceMemoryDiscrete::get_total_concentration(std::string monomer_type, double *phi)
{
    try
    {
        const int M = this->cb->get_total_grid();
        // Initialize array
        for(int i=0; i<M; i++)
            phi[i] = 0.0;

        // For each block
        for(const auto& block: phi_block)
        {
            std::string key_left = std::get<1>(block.first);
            int n_segment_right = this->propagator_computation_optimizer->get_computation_block(block.first).n_segment_right;
            if (PropagatorCode::get_monomer_type_from_key(key_left) == monomer_type && n_segment_right != 0)
            {
                for(int i=0; i<M; i++)
                    phi[i] += block.second[i]; 
            }
        }
        // For each solvent
        for(int s=0;s<this->molecules->get_n_solvent_types();s++)
        {
            if (std::get<1>(this->molecules->get_solvent(s)) == monomer_type)
            {
                double *phi_solvent_ = phi_solvent[s];
                for(int i=0; i<M; i++)
                    phi[i] += phi_solvent_[i];
            }
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaComputationReduceMemoryDiscrete::get_total_concentration(int p, std::string monomer_type, double *phi)
{
    try
    {
        const int M = this->cb->get_total_grid();
        const int P = this->molecules->get_n_polymer_types();

        if (p < 0 || p > P-1)
            throw_with_line_number("Index (" + std::to_string(p) + ") must be in range [0, " + std::to_string(P-1) + "]");

        // Initialize array
        for(int i=0; i<M; i++)
            phi[i] = 0.0;

        // For each block
        for(const auto& block: phi_block)
        {
            int polymer_idx = std::get<0>(block.first);
            std::string key_left = std::get<1>(block.first);
            int n_segment_right = this->propagator_computation_optimizer->get_computation_block(block.first).n_segment_right;
            if (polymer_idx == p && PropagatorCode::get_monomer_type_from_key(key_left) == monomer_type && n_segment_right != 0)
            {
                for(int i=0; i<M; i++)
                    phi[i] += block.second[i]; 
            }
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaComputationReduceMemoryDiscrete::get_total_concentration_gce(double fugacity, int p, std::string monomer_type, double *phi)
{
    try
    {
        const int M = this->cb->get_total_grid();
        const int P = this->molecules->get_n_polymer_types();

        if (p < 0 || p > P-1)
            throw_with_line_number("Index (" + std::to_string(p) + ") must be in range [0, " + std::to_string(P-1) + "]");

        // Initialize array
        for(int i=0; i<M; i++)
            phi[i] = 0.0;

        // For each block
        for(const auto& block: phi_block)
        {
            int polymer_idx = std::get<0>(block.first);
            std::string key_left = std::get<1>(block.first);
            int n_segment_right = this->propagator_computation_optimizer->get_computation_block(block.first).n_segment_right;
            if (polymer_idx == p && PropagatorCode::get_monomer_type_from_key(key_left) == monomer_type && n_segment_right != 0)
            {
                Polymer& pc = this->molecules->get_polymer(p);
                double norm = fugacity/pc.get_volume_fraction()*pc.get_alpha()*this->single_polymer_partitions[p];
                for(int i=0; i<M; i++)
                    phi[i] += block.second[i]*norm; 
            }
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaComputationReduceMemoryDiscrete::get_block_concentration(int p, double *phi)
{
    try
    {
        const int M = this->cb->get_total_grid();
        const int P = this->molecules->get_n_polymer_types();

        if (p < 0 || p > P-1)
            throw_with_line_number("Index (" + std::to_string(p) + ") must be in range [0, " + std::to_string(P-1) + "]");

        if (this->propagator_computation_optimizer->use_aggregation())
            throw_with_line_number("Disable 'aggregation' option to obtain concentration of each block.");

        Polymer& pc = this->molecules->get_polymer(p);
        std::vector<Block>& blocks = pc.get_blocks();

        for(size_t b=0; b<blocks.size(); b++)
        {
            std::string key_left  = pc.get_propagator_key(blocks[b].v, blocks[b].u);
            std::string key_right = pc.get_propagator_key(blocks[b].u, blocks[b].v);
            if (key_left < key_right)
                key_left.swap(key_right);

            double* _essential_phi_block = phi_block[std::make_tuple(p, key_left, key_right)];
            for(int i=0; i<M; i++)
                phi[i+b*M] = _essential_phi_block[i]; 
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
double CudaComputationReduceMemoryDiscrete::get_solvent_partition(int s)
{
    try
    {
        return this->single_solvent_partitions[s];
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaComputationReduceMemoryDiscrete::get_solvent_concentration(int s, double *phi)
{
    try
    {
        const int M = this->cb->get_total_grid();
        const int S = this->molecules->get_n_solvent_types();

        if (s < 0 || s > S-1)
            throw_with_line_number("Index (" + std::to_string(s) + ") must be in range [0, " + std::to_string(S-1) + "]");

        double *phi_solvent_ = phi_solvent[s];
        for(int i=0; i<M; i++)
            phi[i] = phi_solvent_[i];
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaComputationReduceMemoryDiscrete::compute_stress()
{
    // This method should be invoked after invoking compute_statistics().

    // To calculate stress, we multiply weighted fourier basis to q(k)*q^dagger(-k).
    // We only need the real part of stress calculation.
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();
        const int N_GPUS = CudaCommon::get_instance().get_n_gpus();

        const int DIM = this->cb->get_dim();
        const int M   = this->cb->get_total_grid();

        std::map<std::tuple<int, std::string, std::string>, std::array<double,3>> block_dq_dl[n_streams];

        // Reset stress map
        for(const auto& item: phi_block)
        {
            for(int i=0; i<n_streams; i++)
                for(int d=0; d<3; d++)
                    block_dq_dl[i][item.first][d] = 0.0;
        }

        // Compute stress for each block
        #pragma omp parallel for num_threads(n_streams)
        for(size_t b=0; b<phi_block.size();b++)
        {
            const int STREAM = omp_get_thread_num();
            int gpu = omp_get_thread_num() % N_GPUS;
            gpu_error_check(hipSetDevice(gpu));

            auto block = phi_block.begin();
            advance(block, b);
            const auto& key   = block->first;

            int p                 = std::get<0>(key);
            std::string key_left  = std::get<1>(key);
            std::string key_right = std::get<2>(key);

            const int N_RIGHT        = this->propagator_computation_optimizer->get_computation_block(key).n_segment_right;
            const int N_LEFT         = this->propagator_computation_optimizer->get_computation_block(key).n_segment_left;
            std::string monomer_type = this->propagator_computation_optimizer->get_computation_block(key).monomer_type;
            int n_repeated           = this->propagator_computation_optimizer->get_computation_block(key).n_repeated;

            // If there is no segment
            if(N_RIGHT == 0)
                continue;

            double **q_1 = propagator[key_left];      // Propagator q
            double **q_2 = propagator[key_right];     // Propagator q^dagger

            std::array<double,3> _block_dq_dl = {0.0, 0.0, 0.0};

            // Check block_stress_computation_plan
            const auto& _block_stress_compuation_key = block_stress_computation_plan[key];
            if(_block_stress_compuation_key.size() != (unsigned int) (N_RIGHT+1))
            {
                throw_with_line_number("Mismatch of block_stress_computation_plan("
                    + std::to_string(p) + "," + key_left + "," + key_right + ") "
                    + std::to_string(_block_stress_compuation_key.size()) + ".size() with N+1 (" + std::to_string(N_RIGHT+1) + ")");
            }

            // Variables for block_stress_computation_plan
            double *propagator_left;
            double *propagator_right;

            double *d_segment_stress;
            double segment_stress[DIM];
            gpu_error_check(hipMalloc((void**)&d_segment_stress, sizeof(double)*3));

            int prev, next;
            prev = 0;
            next = 1;

            // Create events
            hipEvent_t kernel_done;
            hipEvent_t memcpy_done;
            gpu_error_check(hipEventCreate(&kernel_done));
            gpu_error_check(hipEventCreate(&memcpy_done));

            // Copy memory from device to device
            propagator_left  = std::get<0>(_block_stress_compuation_key[0]);
            propagator_right = std::get<1>(_block_stress_compuation_key[0]);

            if (propagator_left != nullptr)
            {
                gpu_error_check(hipMemcpyAsync(&d_q_pair[STREAM][prev][0], propagator_left,  sizeof(double)*M, hipMemcpyHostToDevice, streams[STREAM][1]));
                gpu_error_check(hipMemcpyAsync(&d_q_pair[STREAM][prev][M], propagator_right, sizeof(double)*M, hipMemcpyHostToDevice, streams[STREAM][1]));
                gpu_error_check(hipEventRecord(memcpy_done, streams[STREAM][1]));
            }
            gpu_error_check(hipStreamWaitEvent(streams[STREAM][0], memcpy_done, 0));

            for(int n=0; n<=N_RIGHT; n++)
            {
                // STREAM 1: copy memory from device to device
                if (n+1 <= N_RIGHT)
                {
                    propagator_left  = std::get<0>(_block_stress_compuation_key[n+1]);
                    propagator_right = std::get<1>(_block_stress_compuation_key[n+1]);

                    if (propagator_left != nullptr)
                    {
                        gpu_error_check(hipMemcpyAsync(&d_q_pair[STREAM][next][0], propagator_left,  sizeof(double)*M, hipMemcpyHostToDevice, streams[STREAM][1]));
                        gpu_error_check(hipMemcpyAsync(&d_q_pair[STREAM][next][M], propagator_right, sizeof(double)*M, hipMemcpyHostToDevice, streams[STREAM][1]));
                        gpu_error_check(hipEventRecord(memcpy_done, streams[STREAM][1]));
                    }
                }

                // STREAM 0: Compute stress
                propagator_left = std::get<0>(_block_stress_compuation_key[n]);
                bool is_half_bond_length = std::get<2>(_block_stress_compuation_key[n]);
                if (propagator_left != nullptr)
                {
                    propagator_solver->compute_single_segment_stress(
                        gpu, STREAM, d_q_pair[STREAM][prev], d_segment_stress,
                        monomer_type, is_half_bond_length);
                    gpu_error_check(hipEventRecord(kernel_done, streams[STREAM][0]));
                }

                // Wait until computation and memory copy are done
                gpu_error_check(hipStreamWaitEvent(streams[STREAM][1], kernel_done, 0));
                gpu_error_check(hipStreamWaitEvent(streams[STREAM][0], memcpy_done, 0));

                if (propagator_left != nullptr)
                {
                    gpu_error_check(hipMemcpy(segment_stress, d_segment_stress, sizeof(double)*DIM, hipMemcpyDeviceToHost));
                    for(int d=0; d<DIM; d++)
                        _block_dq_dl[d] += segment_stress[d]*n_repeated;
                }
                std::swap(prev, next);
            }
            gpu_error_check(hipStreamSynchronize(streams[STREAM][0]));
            gpu_error_check(hipStreamSynchronize(streams[STREAM][1]));
            gpu_error_check(hipEventDestroy(kernel_done));
            gpu_error_check(hipEventDestroy(memcpy_done));

            // Copy stress data
            for(int d=0; d<DIM; d++)
                block_dq_dl[STREAM][key][d] += _block_dq_dl[d];
                
            hipFree(d_segment_stress);
        }
        // Synchronize all GPUs
        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            gpu_error_check(hipDeviceSynchronize());
        }
        gpu_error_check(hipSetDevice(0));
        
        // Compute total stress
        int n_polymer_types = this->molecules->get_n_polymer_types();
        for(int p=0; p<n_polymer_types; p++)
            for(int d=0; d<DIM; d++)
                this->dq_dl[p][d] = 0.0;
        for(const auto& block: phi_block)
        {
            const auto& key       = block.first;
            int p                 = std::get<0>(key);
            std::string key_left  = std::get<1>(key);
            std::string key_right = std::get<2>(key);
            Polymer& pc = this->molecules->get_polymer(p);

            for(int i=0; i<n_streams; i++)
                for(int d=0; d<DIM; d++)
                    this->dq_dl[p][d] += block_dq_dl[i][key][d];
        }
        for(int p=0; p<n_polymer_types; p++){
            for(int d=0; d<DIM; d++)
                this->dq_dl[p][d] /= -3.0*this->cb->get_lx(d)*M*M/this->molecules->get_ds();
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaComputationReduceMemoryDiscrete::get_chain_propagator(double *q_out, int polymer, int v, int u, int n)
{ 
    // This method should be invoked after invoking compute_statistics()

    // Get chain propagator for a selected polymer, block and direction.
    // This is made for debugging and testing.
    try
    {
        const int M = this->cb->get_total_grid();
        Polymer& pc = this->molecules->get_polymer(polymer);
        std::string dep = pc.get_propagator_key(v,u);

        if (this->propagator_computation_optimizer->get_computation_propagators().find(dep) == this->propagator_computation_optimizer->get_computation_propagators().end())
            throw_with_line_number("Could not find the propagator code '" + dep + "'. Disable 'aggregation' option to obtain propagator_computation_optimizer.");
            
        const int N_RIGHT = this->propagator_computation_optimizer->get_computation_propagator(dep).max_n_segment;
        if (n < 1 || n > N_RIGHT)
            throw_with_line_number("n (" + std::to_string(n) + ") must be in range [1, " + std::to_string(N_RIGHT) + "]");

        double* _propagator = propagator[dep][n];
        for(int i=0; i<M; i++)
            q_out[i] = _propagator[i];
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
bool CudaComputationReduceMemoryDiscrete::check_total_partition()
{
    const int M = this->cb->get_total_grid();
    int n_polymer_types = this->molecules->get_n_polymer_types();
    std::vector<std::vector<double>> total_partitions;
    for(int p=0;p<n_polymer_types;p++)
    {
        std::vector<double> total_partitions_p;
        total_partitions.push_back(total_partitions_p);
    }

    gpu_error_check(hipSetDevice(0));
    for(const auto& block: phi_block)
    {
        const auto& key = block.first;
        int p                 = std::get<0>(key);
        std::string key_left  = std::get<1>(key);
        std::string key_right = std::get<2>(key);

        int n_segment_right = this->propagator_computation_optimizer->get_computation_block(key).n_segment_right;
        int n_segment_left  = this->propagator_computation_optimizer->get_computation_block(key).n_segment_left;
        int n_repeated      = this->propagator_computation_optimizer->get_computation_block(key).n_repeated;
        int n_propagators   = this->propagator_computation_optimizer->get_computation_block(key).v_u.size();

        std::string monomer_type = this->propagator_computation_optimizer->get_computation_block(key).monomer_type;
        double *_d_exp_dw = propagator_solver->d_exp_dw[0][monomer_type];

        #ifndef NDEBUG
        std::cout<< p << ", " << key_left << ", " << key_right << ": " << n_segment_left << ", " << n_segment_right << ", " << n_propagators << ", " << this->propagator_computation_optimizer->get_computation_block(key).n_repeated << std::endl;
        #endif
        
        for(int n=1;n<=n_segment_right;n++)
        {
            // Copy propagators from host to device
            gpu_error_check(hipMemcpy(d_q_block_v[0], propagator[key_left][n_segment_left-n+1], sizeof(double)*M, hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_q_block_u[0], propagator[key_right][n], sizeof(double)*M, hipMemcpyHostToDevice));

            double total_partition = this->cb->inner_product_inverse_weight_device(
                d_q_block_v[0],  // q
                d_q_block_u[0],  // q^dagger
                _d_exp_dw)*n_repeated/this->cb->get_volume();
            
            total_partition /= n_propagators;
            total_partitions[p].push_back(total_partition);

            #ifndef NDEBUG
            std::cout<< p << ", " << n << ": " << total_partition << std::endl;
            #endif
        }
    }

    // Find minimum and maximum of total_partitions
    std::cout<< "Polymer id: maximum,  minimum, and difference of total partitions" << std::endl;
    for(size_t p=0;p<total_partitions.size();p++)
    {
        double max_partition = -1e20;
        double min_partition =  1e20;
        for(size_t n=0;n<total_partitions[p].size();n++)
        {
            if (total_partitions[p][n] > max_partition)
                max_partition = total_partitions[p][n];
            if (total_partitions[p][n] < min_partition)
                min_partition = total_partitions[p][n];
        }
        double diff_partition = std::abs(max_partition - min_partition);

        std::cout<< "\t" << p << ": " << max_partition << ", " << min_partition << ", " << diff_partition << std::endl;
        if (diff_partition > 1e-7)
            return false;
    }
    return true;
}