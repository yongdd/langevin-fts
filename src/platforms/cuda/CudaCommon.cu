#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <string>

#include "CudaCommon.h"

void throw_on_cuda_error(hipError_t code, const char *file, int line, const char *func)
{
    if (code != hipSuccess){
        std::string file_and_line("File: \"" + std::string(file) + "\", line: " + std::to_string(line) + ", function <" + std::string(func) + ">");
        throw thrust::system_error(code, thrust::cuda_category(), file_and_line);
    }
}

CudaCommon::CudaCommon()
{
    try{
        // Intialize NUM_BLOCKS and NUM_THREADS
        const char *ENV_N_BLOCKS  = getenv("LFTS_GPU_NUM_BLOCKS");
        const char *ENV_N_THREADS = getenv("LFTS_GPU_NUM_THREADS");

        std::string env_var_n_blocks (ENV_N_BLOCKS  ? ENV_N_BLOCKS  : "");
        std::string env_var_n_threads(ENV_N_THREADS ? ENV_N_THREADS : "");

        if (env_var_n_blocks.empty())
            this->n_blocks = 256;
        else
            this->n_blocks = std::stoi(env_var_n_blocks);

        if (env_var_n_threads.empty())
            this->n_threads = 256;
        else
            this->n_threads = std::stoi(env_var_n_threads);

        // The number of GPUs
        int devices_count;
        gpu_error_check(hipGetDeviceCount(&devices_count));
        const char *ENV_N_GPUS = getenv("LFTS_NUM_GPUS");
        std::string env_var_n_gpus (ENV_N_GPUS  ? ENV_N_GPUS  : "");

        if (env_var_n_gpus.empty())
            n_gpus = 1;
        else
            n_gpus = std::min(std::min(std::stoi(env_var_n_gpus), devices_count), MAX_GPUS);

        // Check if can access peer GPUs
        if (n_gpus > 1)
        {
            int can_access_from_0_to_1;
            int can_access_from_1_to_0;
            gpu_error_check(hipDeviceCanAccessPeer(&can_access_from_0_to_1, 0, 1));
            gpu_error_check(hipDeviceCanAccessPeer(&can_access_from_1_to_0, 1, 0));

            if (can_access_from_0_to_1 == 1 && can_access_from_1_to_0 == 1)
            {
                gpu_error_check(hipSetDevice(0));
                gpu_error_check(hipDeviceEnablePeerAccess(1, 0));
                gpu_error_check(hipSetDevice(1));
                gpu_error_check(hipDeviceEnablePeerAccess(0, 0));
            }
            else
            {
                std::cout << "Could not establish peer access between GPUs." << std::endl;
                std::cout << "Only one GPU will be used." << std::endl;
                n_gpus = 1;
            }
        }
        gpu_error_check(hipSetDevice(0));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaCommon::set(int n_blocks, int n_threads, int process_idx)
{
    int devices_count;

    this->set_n_blocks(n_blocks);
    this->set_n_threads(n_threads);

    // Change GPU setting
    gpu_error_check(hipGetDeviceCount(&devices_count));
    gpu_error_check(hipSetDevice(process_idx%devices_count));
}
int CudaCommon::get_n_blocks()
{
    return n_blocks;
}
int CudaCommon::get_n_threads()
{
    return n_threads;
}
int CudaCommon::get_n_gpus()
{
    return n_gpus;
}
void CudaCommon::set_n_blocks(int n_blocks)
{
    this->n_blocks = n_blocks;
}
void CudaCommon::set_n_threads(int n_threads)
{
    this->n_threads = n_threads;
}
void CudaCommon::set_idx(int process_idx)
{
    int devices_count;

    // Change GPU setting
    gpu_error_check(hipGetDeviceCount(&devices_count));
    gpu_error_check(hipSetDevice(process_idx%devices_count));
}

__global__ void linear_scaling_real(
            double* dst, const double* src, double a, double b, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = a*src[i] + b;
        i += blockDim.x * gridDim.x;
    }
}

__global__ void exp_real(double* dst,
                        const double* src,
                        double a, 
                        double exp_b, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = a * exp(exp_b*src[i]);
        i += blockDim.x * gridDim.x;
    }
}

__global__ void exp_complex(ftsComplex* dst,
                            const ftsComplex* src,
                            double a, 
                            double exp_b, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i].x = a * exp(exp_b * src[i].x) * cos(exp_b * src[i].y);
        dst[i].y = a * exp(exp_b * src[i].x) * sin(exp_b * src[i].y);
        i += blockDim.x * gridDim.x;
    }
}
__global__ void multi_real(double* dst,
                          const double* src1,
                          const double* src2,
                          double  a, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = a * src1[i] * src2[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void mutiple_multi_real(int n_comp,
                          double* dst,
                          const double* src1,
                          const double* src2,
                          double  a, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {  
        dst[i] = a * src1[i] * src2[i];
        for(int n = 1; n < n_comp; n++)
            dst[i] += a * src1[i+n*M] * src2[i+n*M];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void divide_real(double* dst,
                          const double* src1,
                          const double* src2,
                          double  a, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = a * src1[i]/src2[i];
        i += blockDim.x * gridDim.x;
    }
}
__global__ void add_multi_real(double* dst,
                             const double* src1,
                             const double* src2,
                             double  a, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = dst[i] + a * src1[i] * src2[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void lin_comb(double* dst,
                        double a,
                        const double* src1,
                        double b,
                        const double* src2,
                        const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = a*src1[i] + b*src2[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void add_lin_comb(double* dst,
                           double a,
                           const double* src1,
                           double b,
                           const double* src2,
                           const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = dst[i] + a*src1[i] + b*src2[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void multi_complex_real(ftsComplex* dst,
                                   const double* src, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i].x = dst[i].x * src[i];
        dst[i].y = dst[i].y * src[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void multi_complex_real(ftsComplex* dst,
                                  const double* src, double a, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i].x = a * dst[i].x * src[i];
        dst[i].y = a * dst[i].y * src[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void multi_complex_conjugate(double* dst,
                                 const ftsComplex* src1,
                                 const ftsComplex* src2, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = src1[i].x * src2[i].x + src1[i].y * src2[i].y;
        i += blockDim.x * gridDim.x;
    }
}

__global__ void real_multi_exp_dw_two(
                        double* dst1, const double* src1, const double* exp_dw1,
                        double* dst2, const double* src2, const double* exp_dw2,
                        double  a, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst1[i] = a * src1[i] * exp_dw1[i];
        dst2[i] = a * src2[i] * exp_dw2[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void real_multi_exp_dw_four(
                        double* dst1, const double* src1, const double* exp_dw1,
                        double* dst2, const double* src2, const double* exp_dw2,
                        double* dst3, const double* src3, const double* exp_dw3,
                        double* dst4, const double* src4, const double* exp_dw4,
                        double  a, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst1[i] = a * src1[i] * exp_dw1[i];
        dst2[i] = a * src2[i] * exp_dw2[i];
        dst3[i] = a * src3[i] * exp_dw3[i];
        dst4[i] = a * src4[i] * exp_dw4[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void complex_real_multi_bond_two(
                        ftsComplex* dst1, const double* boltz_bond1,
                        ftsComplex* dst2, const double* boltz_bond2,
                        const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst1[i].x = dst1[i].x * boltz_bond1[i];
        dst1[i].y = dst1[i].y * boltz_bond1[i];
        dst2[i].x = dst2[i].x * boltz_bond2[i];
        dst2[i].y = dst2[i].y * boltz_bond2[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void complex_real_multi_bond_four(
                        ftsComplex* dst1, const double* boltz_bond1,
                        ftsComplex* dst2, const double* boltz_bond2,
                        ftsComplex* dst3, const double* boltz_bond3,
                        ftsComplex* dst4, const double* boltz_bond4,
                        const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst1[i].x = dst1[i].x * boltz_bond1[i];
        dst1[i].y = dst1[i].y * boltz_bond1[i];
        dst2[i].x = dst2[i].x * boltz_bond2[i];
        dst2[i].y = dst2[i].y * boltz_bond2[i];
        dst3[i].x = dst3[i].x * boltz_bond3[i];
        dst3[i].y = dst3[i].y * boltz_bond3[i];
        dst4[i].x = dst4[i].x * boltz_bond4[i];
        dst4[i].y = dst4[i].y * boltz_bond4[i];
        i += blockDim.x * gridDim.x;
    }
}