#include "hip/hip_runtime.h"
#include <complex>
#include <omp.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>

#include "CudaComputationContinuous.h"
#include "CudaComputationBox.h"
#include "CudaSolverPseudoContinuous.h"
#include "CudaSolverRealSpace.h"
#include "SimpsonRule.h"

template <typename T>
CudaComputationContinuous<T>::CudaComputationContinuous(
    ComputationBox<T>* cb,
    Molecules *molecules,
    PropagatorComputationOptimizer *propagator_computation_optimizer,
    std::string method)
    : PropagatorComputation<T>(cb, molecules, propagator_computation_optimizer)
{
    try{
        #ifndef NDEBUG
        std::cout << "--------- Continuous Chain Solver, GPU Version ---------" << std::endl;
        #endif

        const int M = this->cb->get_total_grid();

        // The number of parallel streams for propagator computation
        const char *ENV_OMP_NUM_THREADS = getenv("OMP_NUM_THREADS");
        std::string env_omp_num_threads(ENV_OMP_NUM_THREADS ? ENV_OMP_NUM_THREADS  : "");
        if (env_omp_num_threads.empty())
            n_streams = MAX_STREAMS;
        else
            n_streams =  std::min(std::stoi(env_omp_num_threads), MAX_STREAMS);
        #ifndef NDEBUG
        std::cout << "The number of CPU threads: " << n_streams << std::endl;
        #endif

        // Copy streams
        for(int i=0; i<n_streams; i++)
        {
            gpu_error_check(hipStreamCreate(&streams[i][0])); // for kernel execution
            gpu_error_check(hipStreamCreate(&streams[i][1])); // for memcpy
        }

        this->method = method;
        if(method == "pseudospectral")
            this->propagator_solver = new CudaSolverPseudoContinuous<T>(cb, molecules, n_streams, streams, false);
        else if(method == "realspace")
        {
            if constexpr (std::is_same<T, double>::value) 
                this->propagator_solver = new CudaSolverRealSpace(cb, molecules, n_streams, streams, false);
            else
                throw_with_line_number("Currently, the realspace method is only available for double precision.");
        }

        // Allocate memory for propagators
        if( this->propagator_computation_optimizer->get_computation_propagators().size() == 0)
            throw_with_line_number("There is no propagator code. Add polymers first.");
        for(const auto& item: this->propagator_computation_optimizer->get_computation_propagators())
        {
            std::string key = item.first;
            int max_n_segment = item.second.max_n_segment+1;
            
            propagator_size[key] = max_n_segment;
            d_propagator[key] = new CuDeviceData<T>*[max_n_segment];
            for(int i=0; i<propagator_size[key]; i++)
                gpu_error_check(hipMalloc((void**)&d_propagator[key][i], sizeof(T)*M));

            #ifndef NDEBUG
            propagator_finished[key] = new bool[max_n_segment];
            for(int i=0; i<max_n_segment;i++)
                propagator_finished[key][i] = false;
            #endif
        }

        // Allocate memory for concentrations
        if( this->propagator_computation_optimizer->get_computation_blocks().size() == 0)
            throw_with_line_number("There is no block. Add polymers first.");
        for(const auto& item: this->propagator_computation_optimizer->get_computation_blocks())
        {
            d_phi_block[item.first] = nullptr;
            gpu_error_check(hipMalloc((void**)&d_phi_block[item.first], sizeof(T)*M));
        }

        // Remember one segment for each polymer chain to compute total partition function
        int current_p = 0;
        for(const auto& block: d_phi_block)
        {
            const auto& key = block.first;
            int p                 = std::get<0>(key);
            std::string key_left  = std::get<1>(key);
            std::string key_right = std::get<2>(key);

            // Skip if already found one segment
            if (p != current_p)
                continue;

            int n_aggregated   = this->propagator_computation_optimizer->get_computation_block(key).v_u.size()/
                                 this->propagator_computation_optimizer->get_computation_block(key).n_repeated;
            int n_segment_left = this->propagator_computation_optimizer->get_computation_block(key).n_segment_left;

            single_partition_segment.push_back(std::make_tuple(
                p,
                d_propagator[key_left][n_segment_left],   // q
                d_propagator[key_right][0],               // q_dagger
                n_aggregated                              // how many propagators are aggregated
                ));
            current_p++;
        }

        // Concentrations for each solvent
        for(int s=0;s<this->molecules->get_n_solvent_types();s++)
        {
            CuDeviceData<T> *d_phi_;
            gpu_error_check(hipMalloc((void**)&d_phi_, sizeof(T)*M));
            d_phi_solvent.push_back(d_phi_);
        }

        // Create scheduler for computation of propagator
        sc = new Scheduler(this->propagator_computation_optimizer->get_computation_propagators(), n_streams); 

        // Allocate memory for pseudo-spectral: advance_propagator()
        gpu_error_check(hipMalloc((void**)&d_q_unity, sizeof(T)*M));
        for(int i=0; i<M; i++)
        {
            CuDeviceData<T> q_unity;
            if constexpr (std::is_same<T, double>::value)
                q_unity = 1.0;
            else
                q_unity = make_hipDoubleComplex(1.0, 0.0);
            gpu_error_check(hipMemcpy(&d_q_unity[i], &q_unity, sizeof(T), hipMemcpyHostToDevice));
        }

        // Copy mask to d_q_mask
        if (this->cb->get_mask() != nullptr)
        {
            gpu_error_check(hipMalloc((void**)&d_q_mask, sizeof(double)*M));
            gpu_error_check(hipMemcpy(d_q_mask, this->cb->get_mask(), sizeof(double)*M, hipMemcpyHostToDevice));
        }
        else
            d_q_mask = nullptr;
        gpu_error_check(hipMalloc((void**)&d_phi, sizeof(T)*M));

        // Allocate memory for stress calculation: compute_stress()
        for(int i=0; i<n_streams; i++)
        {
            gpu_error_check(hipMalloc((void**)&d_q_pair[i][0], sizeof(T)*2*M)); // prev
            gpu_error_check(hipMalloc((void**)&d_q_pair[i][1], sizeof(T)*2*M)); // next
        }

        propagator_solver->update_laplacian_operator();
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
template <typename T>
CudaComputationContinuous<T>::~CudaComputationContinuous()
{
    delete propagator_solver;
    delete sc;

    for(const auto& item: d_propagator)
    {
        for(int i=0; i<propagator_size[item.first]; i++)
            hipFree(item.second[i]);
        delete[] item.second;
    }
    for(const auto& item: d_phi_block)
        hipFree(item.second);
    for(const auto& item: d_phi_solvent)
        hipFree(item);

    #ifndef NDEBUG
    for(const auto& item: propagator_finished)
        delete[] item.second;
    #endif

    hipFree(d_phi);

    // For pseudo-spectral: advance_propagator()
    if (d_q_mask != nullptr)
        hipFree(d_q_mask);
    hipFree(d_q_unity);


    // For stress calculation: compute_stress()
    for(int i=0; i<n_streams; i++)
    {
        hipFree(d_q_pair[i][0]);
        hipFree(d_q_pair[i][1]);
    }

    // Destroy streams
    for(int i=0; i<n_streams; i++)
    {
        hipStreamDestroy(streams[i][0]);
        hipStreamDestroy(streams[i][1]);
    }
}
template <typename T>
void CudaComputationContinuous<T>::update_laplacian_operator()
{
    try{
        propagator_solver->update_laplacian_operator();
    }
    catch(std::exception& exc)
    {
        throw_with_line_number(exc.what());
    }
}
template <typename T>
void CudaComputationContinuous<T>::compute_statistics(
    std::map<std::string, const T*> w_input,
    std::map<std::string, const T*> q_init)
{
    this->compute_propagators(w_input, q_init);
    this->compute_concentrations();
}
template <typename T>
void CudaComputationContinuous<T>::compute_propagators(
    std::map<std::string, const T*> w_input,
    std::map<std::string, const T*> q_init)
{
    try{
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = this->cb->get_total_grid();
        const double ds = this->molecules->get_ds();

        std::string device = "cpu";
        hipMemcpyKind cudaMemcpyInputToDevice;
        if (device == "gpu")
            cudaMemcpyInputToDevice = hipMemcpyDeviceToDevice;
        else if(device == "cpu")
            cudaMemcpyInputToDevice = hipMemcpyHostToDevice;
        else
        {
            throw_with_line_number("Invalid device \"" + device + "\".");
        }

        for(const auto& item: this->propagator_computation_optimizer->get_computation_propagators())
        {
            if( w_input.find(item.second.monomer_type) == w_input.end())
                throw_with_line_number("monomer_type \"" + item.second.monomer_type + "\" is not in w_input.");
        }

        // Update dw or d_exp_dw
        propagator_solver->update_dw(device, w_input);
       
        // For each time span
        #ifndef NDEBUG
        for(const auto& item: this->propagator_computation_optimizer->get_computation_propagators())
        {
            std::string key = item.first;
            int max_n_segment = item.second.max_n_segment+1;
            for(int i=0; i<max_n_segment;i++)
                propagator_finished[key][i] = false;
        }
        #endif

        auto& branch_schedule = sc->get_schedule();
        for (auto parallel_job = branch_schedule.begin(); parallel_job != branch_schedule.end(); parallel_job++)
        {
            // For each propagator
            #pragma omp parallel for num_threads(n_streams)
            for(size_t job=0; job<parallel_job->size(); job++)
            {
                const int STREAM = omp_get_thread_num();
                // printf("gpu, STREAM: %d, %d\n ", gpu, STREAM);

                auto& key = std::get<0>((*parallel_job)[job]);
                int n_segment_from = std::get<1>((*parallel_job)[job]);
                int n_segment_to = std::get<2>((*parallel_job)[job]);
                auto& deps = this->propagator_computation_optimizer->get_computation_propagator(key).deps;
                auto monomer_type = this->propagator_computation_optimizer->get_computation_propagator(key).monomer_type;

                // // Display job info
                // #ifndef NDEBUG
                // std::cout << job << " started" << std::endl;
                // #endif

                // Check key
                #ifndef NDEBUG
                if (d_propagator.find(key) == d_propagator.end())
                    std::cout << "Could not find key '" + key + "'. " << std::endl;
                #endif

                CuDeviceData<T> **_d_propagator = d_propagator[key];

                // If it is leaf node
                if(n_segment_from == 0 && deps.size() == 0)
                {
                     // q_init
                    if (key[0] == '{')
                    {
                        std::string g = PropagatorCode::get_q_input_idx_from_key(key);
                        if (q_init.find(g) == q_init.end())
                            std::cout << "Could not find q_init[\"" + g + "\"]." << std::endl;
                        gpu_error_check(hipMemcpyAsync(_d_propagator[0], q_init[g],
                            sizeof(T)*M, cudaMemcpyInputToDevice, streams[STREAM][0]));
                    }
                    else
                    {
                        gpu_error_check(hipMemcpyAsync(_d_propagator[0], d_q_unity,
                            sizeof(T)*M, hipMemcpyDeviceToDevice, streams[STREAM][0]));
                    }

                    #ifndef NDEBUG
                    propagator_finished[key][0] = true;
                    #endif
                }
                // If it is not leaf node
                else if (n_segment_from == 0 && deps.size() > 0)
                {
                    // If it is aggregated
                    if (key[0] == '[')
                    {
                        // Initialize to zero
                        gpu_error_check(hipMemsetAsync(_d_propagator[0], 0, sizeof(T)*M, streams[STREAM][0]));

                        // Add all propagators at junction if necessary 
                        for(size_t d=0; d<deps.size(); d++)
                        {
                            std::string sub_dep = std::get<0>(deps[d]);
                            int sub_n_segment   = std::get<1>(deps[d]);
                            int sub_n_repeated  = std::get<2>(deps[d]);

                            // Check sub key
                            #ifndef NDEBUG
                            if (d_propagator.find(sub_dep) == d_propagator.end())
                                std::cout << "Could not find sub key '" + sub_dep + "'. " << std::endl;
                            if (!propagator_finished[sub_dep][sub_n_segment])
                                std::cout << "Could not compute '" + key +  "', since '"+ sub_dep + std::to_string(sub_n_segment) + "' is not prepared." << std::endl;
                            #endif

                            ker_lin_comb<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
                                _d_propagator[0], 1.0, _d_propagator[0],
                                sub_n_repeated, d_propagator[sub_dep][sub_n_segment], M);
                        }

                        #ifndef NDEBUG
                        propagator_finished[key][0] = true;
                        #endif
                    }
                    else
                    {
                        // Initialize to one
                        gpu_error_check(hipMemcpyAsync(_d_propagator[0], d_q_unity,
                            sizeof(T)*M, hipMemcpyDeviceToDevice, streams[STREAM][0]));

                        // Multiply all propagators at junction if necessary 
                        for(size_t d=0; d<deps.size(); d++)
                        {
                            std::string sub_dep = std::get<0>(deps[d]);
                            int sub_n_segment   = std::get<1>(deps[d]);

                            // Check sub key
                            #ifndef NDEBUG
                            if (d_propagator.find(sub_dep) == d_propagator.end())
                                std::cout << "Could not find sub key '" + sub_dep + "'. " << std::endl;
                            if (!propagator_finished[sub_dep][sub_n_segment])
                                std::cout << "Could not compute '" + key +  "', since '"+ sub_dep + std::to_string(sub_n_segment) + "' is not prepared." << std::endl;
                            #endif

                            ker_multi<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(
                                _d_propagator[0], _d_propagator[0],
                                d_propagator[sub_dep][sub_n_segment], 1.0, M);
                        }
                        
                        #ifndef NDEBUG
                        propagator_finished[key][0] = true;
                        #endif
                    }
                }

                // Multiply mask
                if (n_segment_from == 0 && d_q_mask != nullptr)
                    ker_multi<<<N_BLOCKS, N_THREADS, 0, streams[STREAM][0]>>>(_d_propagator[0], _d_propagator[0], d_q_mask, 1.0, M);

                for(int n=n_segment_from; n<n_segment_to; n++)
                {
                    #ifndef NDEBUG
                    if (!propagator_finished[key][n])
                        std::cout << "unfinished, key: " + key + ", " + std::to_string(n) << std::endl;
                    if (propagator_finished[key][n+1])
                        std::cout << "already finished: " + key + ", " + std::to_string(n+1) << std::endl;
                    #endif

                    // STREAM 0
                    propagator_solver->advance_propagator(
                        STREAM, 
                        _d_propagator[n],
                        _d_propagator[n+1],
                        monomer_type, d_q_mask);

                    #ifndef NDEBUG
                    propagator_finished[key][n+1] = true;
                    #endif
                }

                gpu_error_check(hipStreamSynchronize(streams[STREAM][0]));
                gpu_error_check(hipStreamSynchronize(streams[STREAM][1]));

                // // Display job info
                // #ifndef NDEBUG
                // std::cout << job << " finished" << std::endl;
                // #endif
            }
            gpu_error_check(hipDeviceSynchronize());
        }

        // Compute total partition function of each distinct polymers
        for(const auto& segment_info: single_partition_segment)
        {
            int p                 = std::get<0>(segment_info);
            CuDeviceData<T> *d_propagator_left  = std::get<1>(segment_info);
            CuDeviceData<T> *d_propagator_right = std::get<2>(segment_info);
            int n_aggregated      = std::get<3>(segment_info);

            this->single_polymer_partitions[p] = dynamic_cast<CudaComputationBox<T>*>(this->cb)->inner_product_device(d_propagator_left, d_propagator_right)
                /(n_aggregated*this->cb->get_volume());
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
template <typename T>
void CudaComputationContinuous<T>::compute_concentrations()
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();
        const int M = this->cb->get_total_grid();

        // Calculate segment concentrations
        for(const auto& d_block: d_phi_block)
        {
            const auto& key = d_block.first;
            int p                 = std::get<0>(key);
            std::string key_left  = std::get<1>(key);
            std::string key_right = std::get<2>(key);

            int n_segment_right = this->propagator_computation_optimizer->get_computation_block(key).n_segment_right;
            int n_segment_left  = this->propagator_computation_optimizer->get_computation_block(key).n_segment_left;
            int n_repeated = this->propagator_computation_optimizer->get_computation_block(key).n_repeated;

            // If there is no segment
            if(n_segment_right == 0)
            {
                gpu_error_check(hipMemset(d_block.second, 0, sizeof(T)*M));
                continue;
            }

            // Check keys
            #ifndef NDEBUG
            if (d_propagator.find(key_left) == d_propagator.end())
                throw_with_line_number("Could not find key_left key'" + key_left + "'. ");
            if (d_propagator.find(key_right) == d_propagator.end())
                throw_with_line_number("Could not find key_right key'" + key_right + "'. ");
            #endif

            // Calculate phi of one block (possibly multiple blocks when using aggregation)
            calculate_phi_one_block(
                d_block.second,           // phi
                d_propagator[key_left],   // dependency v
                d_propagator[key_right],  // dependency u
                n_segment_right,
                n_segment_left);

            // Normalize concentration
            Polymer& pc = this->molecules->get_polymer(p);

            T _norm = (this->molecules->get_ds()*pc.get_volume_fraction()/pc.get_alpha()*n_repeated)/this->single_polymer_partitions[p];
            CuDeviceData<T> norm;
            if constexpr (std::is_same<T, double>::value)
                norm = _norm;
            else
                norm = stdToCuDoubleComplex(_norm);
            ker_lin_comb<<<N_BLOCKS, N_THREADS>>>(d_block.second, norm, d_block.second, 0.0, d_block.second, M);
        }

        // Calculate partition functions and concentrations of solvents
        for(int s=0; s<this->molecules->get_n_solvent_types(); s++)
        {
            CuDeviceData<T> *d_phi_ = d_phi_solvent[s];
            double volume_fraction   = std::get<0>(this->molecules->get_solvent(s));
            std::string monomer_type = std::get<1>(this->molecules->get_solvent(s));
            CuDeviceData<T> *_d_exp_dw = propagator_solver->d_exp_dw[monomer_type];

            this->single_solvent_partitions[s] = dynamic_cast<CudaComputationBox<T>*>(this->cb)->inner_product_device(_d_exp_dw, _d_exp_dw)/this->cb->get_volume();

            CuDeviceData<T> norm;
            if constexpr (std::is_same<T, double>::value)
            {
                norm = volume_fraction;
                norm = norm/this->single_solvent_partitions[s];
            }
            else
            {
                norm = make_hipDoubleComplex(volume_fraction, 0.0);
                norm = hipCdiv(norm, stdToCuDoubleComplex(this->single_solvent_partitions[s]));
            }
            ker_multi<<<N_BLOCKS, N_THREADS>>>(d_phi_,_d_exp_dw, _d_exp_dw, norm, M);
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
template <typename T>
void CudaComputationContinuous<T>::calculate_phi_one_block(
    CuDeviceData<T> *d_phi, CuDeviceData<T> **d_q_1, CuDeviceData<T> **d_q_2, const int N_RIGHT, const int N_LEFT)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = this->cb->get_total_grid();
        std::vector<double> simpson_rule_coeff = SimpsonRule::get_coeff(N_RIGHT);

        // Compute segment concentration
        ker_multi<<<N_BLOCKS, N_THREADS>>>(d_phi, d_q_1[N_LEFT], d_q_2[0], simpson_rule_coeff[0], M);
        for(int n=1; n<=N_RIGHT; n++)
        {
            ker_add_multi<<<N_BLOCKS, N_THREADS>>>(d_phi, d_q_1[N_LEFT-n], d_q_2[n], simpson_rule_coeff[n], M);
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
template <typename T>
T CudaComputationContinuous<T>::get_total_partition(int polymer)
{
    try
    {
        return this->single_polymer_partitions[polymer];
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
template <typename T>
void CudaComputationContinuous<T>::get_total_concentration(std::string monomer_type, T *phi)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();
        const int M = this->cb->get_total_grid();

        // Initialize to zero
        gpu_error_check(hipMemset(d_phi, 0, sizeof(T)*M));

        // For each block
        for(const auto& d_block: d_phi_block)
        {
            const auto& key = d_block.first;
            std::string key_left = std::get<1>(key);
            int n_segment_right = this->propagator_computation_optimizer->get_computation_block(key).n_segment_right;
            if (PropagatorCode::get_monomer_type_from_key(key_left) == monomer_type && n_segment_right != 0)
                ker_lin_comb<<<N_BLOCKS, N_THREADS>>>(d_phi, 1.0, d_phi, 1.0, d_block.second, M);
        }

        // For each solvent
        for(int s=0;s<this->molecules->get_n_solvent_types();s++)
        {
            if (std::get<1>(this->molecules->get_solvent(s)) == monomer_type)
                ker_lin_comb<<<N_BLOCKS, N_THREADS>>>(d_phi, 1.0, d_phi, 1.0, d_phi_solvent[s], M);
        }
        gpu_error_check(hipMemcpy(phi, d_phi, sizeof(T)*M, hipMemcpyDeviceToHost));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
template <typename T>
void CudaComputationContinuous<T>::get_total_concentration(int p, std::string monomer_type, T *phi)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = this->cb->get_total_grid();
        const int P = this->molecules->get_n_polymer_types();

        if (p < 0 || p > P-1)
            throw_with_line_number("Index (" + std::to_string(p) + ") must be in range [0, " + std::to_string(P-1) + "]");

        // Initialize to zero
        gpu_error_check(hipMemset(d_phi, 0, sizeof(T)*M));

        // For each block
        for(const auto& d_block: d_phi_block)
        {
            const auto& key = d_block.first;
            int polymer_idx = std::get<0>(key);
            std::string key_left = std::get<1>(key);
            int n_segment_right = this->propagator_computation_optimizer->get_computation_block(key).n_segment_right;
            if (polymer_idx == p && PropagatorCode::get_monomer_type_from_key(key_left) == monomer_type && n_segment_right != 0)
                ker_lin_comb<<<N_BLOCKS, N_THREADS>>>(d_phi, 1.0, d_phi, 1.0, d_block.second, M);
        }
        gpu_error_check(hipMemcpy(phi, d_phi, sizeof(T)*M, hipMemcpyDeviceToHost));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
template <typename T>
void CudaComputationContinuous<T>::get_total_concentration_gce(double fugacity, int p, std::string monomer_type, T *phi)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = this->cb->get_total_grid();
        const int P = this->molecules->get_n_polymer_types();

        if (p < 0 || p > P-1)
            throw_with_line_number("Index (" + std::to_string(p) + ") must be in range [0, " + std::to_string(P-1) + "]");

        // Initialize to zero
        gpu_error_check(hipMemset(d_phi, 0, sizeof(T)*M));

        // For each block
        for(const auto& d_block: d_phi_block)
        {
            const auto& key = d_block.first;
            int polymer_idx = std::get<0>(key);
            std::string key_left = std::get<1>(key);
            int n_segment_right = this->propagator_computation_optimizer->get_computation_block(key).n_segment_right;
            if (polymer_idx == p && PropagatorCode::get_monomer_type_from_key(key_left) == monomer_type && n_segment_right != 0)
            {
                Polymer& pc = this->molecules->get_polymer(p);

                CuDeviceData<T> norm;
                if constexpr (std::is_same<T, double>::value)
                    norm = fugacity/pc.get_volume_fraction()*pc.get_alpha()*this->single_polymer_partitions[p];
                else
                    norm = stdToCuDoubleComplex(fugacity/pc.get_volume_fraction()*pc.get_alpha()*this->single_polymer_partitions[p]);
                ker_lin_comb<<<N_BLOCKS, N_THREADS>>>(d_phi, norm, d_block.second, 1.0, d_phi, M);
            }
        }
        gpu_error_check(hipMemcpy(phi, d_phi, sizeof(T)*M, hipMemcpyDeviceToHost));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
template <typename T>
void CudaComputationContinuous<T>::get_block_concentration(int p, T *phi)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = this->cb->get_total_grid();
        const int P = this->molecules->get_n_polymer_types();

        if (p < 0 || p > P-1)
            throw_with_line_number("Index (" + std::to_string(p) + ") must be in range [0, " + std::to_string(P-1) + "]");

        if (this->propagator_computation_optimizer->use_aggregation())
            throw_with_line_number("Disable 'aggregation' option to invoke 'get_block_concentration'.");

        // Initialize to zero
        gpu_error_check(hipMemset(d_phi, 0, sizeof(T)*M));

        Polymer& pc = this->molecules->get_polymer(p);
        std::vector<Block>& blocks = pc.get_blocks();

        for(size_t b=0; b<blocks.size(); b++)
        {
            std::string key_left  = pc.get_propagator_key(blocks[b].v, blocks[b].u);
            std::string key_right = pc.get_propagator_key(blocks[b].u, blocks[b].v);
            if (key_left < key_right)
                key_left.swap(key_right);

            ker_lin_comb<<<N_BLOCKS, N_THREADS>>>(d_phi, 0.0, d_phi, 1.0, d_phi_block[std::make_tuple(p, key_left, key_right)], M);
            gpu_error_check(hipMemcpy(&phi[b*M], d_phi, sizeof(T)*M, hipMemcpyDeviceToHost));
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
template <typename T>
T CudaComputationContinuous<T>::get_solvent_partition(int s)
{
    try
    {
        return this->single_solvent_partitions[s];
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
template <typename T>
void CudaComputationContinuous<T>::get_solvent_concentration(int s, T *phi)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = this->cb->get_total_grid();
        const int S = this->molecules->get_n_solvent_types();

        if (s < 0 || s > S-1)
            throw_with_line_number("Index (" + std::to_string(s) + ") must be in range [0, " + std::to_string(S-1) + "]");

        gpu_error_check(hipMemcpy(phi, d_phi_solvent[s], sizeof(T)*M, hipMemcpyDeviceToHost));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
template <typename T>
void CudaComputationContinuous<T>::compute_stress()
{
    // This method should be invoked after invoking compute_statistics().

    // To calculate stress, we multiply weighted fourier basis to q(k)*q^dagger(-k).
    // We only need the real part of stress calculation.

    try
    {
        // if constexpr (std::is_same<T, std::complex<double>>::value)
        //     throw_with_line_number("Currently, stress computation is not suppoted for complex number type.");

        if (this->method == "realspace")
            throw_with_line_number("Currently, the real-space method does not support stress computation.");

        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int DIM = this->cb->get_dim();
        const int M   = this->cb->get_total_grid();

        std::map<std::tuple<int, std::string, std::string>, std::array<T,3>> block_dq_dl[n_streams];

        // Reset stress map
        for(const auto& item: d_phi_block)
        {
            for(int i=0; i<n_streams; i++)
                for(int d=0; d<3; d++)
                    block_dq_dl[i][item.first][d] = 0.0;
        }

        // Compute stress for each block
        #pragma omp parallel for num_threads(n_streams)
        for(size_t b=0; b<d_phi_block.size();b++)
        {
            const int STREAM = omp_get_thread_num();

            auto block = d_phi_block.begin();
            advance(block, b);
            const auto& key   = block->first;

            // printf("start, b, gpu, STREAM: %2d, %2d, %2d\n", b, gpu, STREAM);

            int p                 = std::get<0>(key);
            std::string key_left  = std::get<1>(key);
            std::string key_right = std::get<2>(key);

            const int N_RIGHT = this->propagator_computation_optimizer->get_computation_block(key).n_segment_right;
            const int N_LEFT  = this->propagator_computation_optimizer->get_computation_block(key).n_segment_left;
            std::string monomer_type = this->propagator_computation_optimizer->get_computation_block(key).monomer_type;
            int n_repeated = this->propagator_computation_optimizer->get_computation_block(key).n_repeated;

            // If there is no segment
            if(N_RIGHT == 0)
                continue;

            // std::cout << p << ", " << key_left << ", " << key_right << ", " << N << ", " << N_LEFT << std::endl;

            std::vector<double> s_coeff = SimpsonRule::get_coeff(N_RIGHT);
            CuDeviceData<T>** d_q_1 = d_propagator[key_left];     // dependency v
            CuDeviceData<T>** d_q_2 = d_propagator[key_right];    // dependency u

            std::array<T,3> _block_dq_dl;
            for(int i=0; i<3; i++)
                _block_dq_dl[i] = 0.0;
            
            CuDeviceData<T> *d_segment_stress;
            T segment_stress[DIM];
            gpu_error_check(hipMalloc((void**)&d_segment_stress, sizeof(T)*3));
                
            int prev, next;
            prev = 0;
            next = 1;

            // Create events
            hipEvent_t kernel_done;
            hipEvent_t memcpy_done;
            gpu_error_check(hipEventCreate(&kernel_done));
            gpu_error_check(hipEventCreate(&memcpy_done));

            gpu_error_check(hipMemcpyAsync(&d_q_pair[STREAM][prev][0], d_q_1[N_LEFT],
                    sizeof(T)*M,hipMemcpyDeviceToDevice, streams[STREAM][1]));
            gpu_error_check(hipMemcpyAsync(&d_q_pair[STREAM][prev][M], d_q_2[0],
                    sizeof(T)*M,hipMemcpyDeviceToDevice, streams[STREAM][1]));

            gpu_error_check(hipEventRecord(memcpy_done, streams[STREAM][1]));
            gpu_error_check(hipStreamWaitEvent(streams[STREAM][0], memcpy_done, 0));

            for(int n=0; n<=N_RIGHT; n++)
            {
                // STREAM 1: Copy data
                if (n+1 <= N_RIGHT)
                {
                    gpu_error_check(hipMemcpyAsync(&d_q_pair[STREAM][next][0], d_q_1[N_LEFT-n-1],
                            sizeof(T)*M,hipMemcpyDeviceToDevice, streams[STREAM][1]));
                    gpu_error_check(hipMemcpyAsync(&d_q_pair[STREAM][next][M], d_q_2[n+1],
                            sizeof(T)*M,hipMemcpyDeviceToDevice, streams[STREAM][1]));
                    gpu_error_check(hipEventRecord(memcpy_done, streams[STREAM][1]));
                }

                // STREAM 0: Compute stress
                propagator_solver->compute_single_segment_stress(
                    STREAM, d_q_pair[STREAM][prev], d_segment_stress,
                    monomer_type, false);   
                gpu_error_check(hipEventRecord(kernel_done, streams[STREAM][0]));

                // Wait until computation and memory copy are done
                gpu_error_check(hipStreamWaitEvent(streams[STREAM][1], kernel_done, 0));
                gpu_error_check(hipStreamWaitEvent(streams[STREAM][0], memcpy_done, 0));

                gpu_error_check(hipMemcpy(segment_stress, d_segment_stress, sizeof(T)*DIM, hipMemcpyDeviceToHost));
                for(int d=0; d<DIM; d++)
                    _block_dq_dl[d] += segment_stress[d]*(s_coeff[n]*n_repeated);

                // std::cout << key_left << ", "  << key_right << ", " << n << ", " << segment_stress[0] << ", " << segment_stress[1] << ", " << segment_stress[2] << std::endl;

                std::swap(prev, next);
            }
            gpu_error_check(hipStreamSynchronize(streams[STREAM][0]));
            gpu_error_check(hipStreamSynchronize(streams[STREAM][1]));
            gpu_error_check(hipEventDestroy(kernel_done));
            gpu_error_check(hipEventDestroy(memcpy_done));

            // std::cout << p << ", " << key_left << ", " << key_right << ", " << N << ", " << N_LEFT << std::endl;
            // std::cout << "STREAM, _block_dq_dl[0] " << STREAM  << ", " << _block_dq_dl[0] << std::endl;

            for(int d=0; d<DIM; d++)
                block_dq_dl[STREAM][key][d] += _block_dq_dl[d];

            hipFree(d_segment_stress);
        }
        gpu_error_check(hipDeviceSynchronize());

        // Compute total stress
        int n_polymer_types = this->molecules->get_n_polymer_types();
        for(int p=0; p<n_polymer_types; p++)
            for(int d=0; d<DIM; d++)
                this->dq_dl[p][d] = 0.0;
        for(const auto& d_block: d_phi_block)
        {
            const auto& key       = d_block.first;
            int p                 = std::get<0>(key);
            std::string key_left  = std::get<1>(key);
            std::string key_right = std::get<2>(key);

            for(int i=0; i<n_streams; i++)
                for(int d=0; d<DIM; d++)
                    this->dq_dl[p][d] += block_dq_dl[i][key][d];
        }
        for(int p=0; p<n_polymer_types; p++)
        {
            for(int d=0; d<DIM; d++)
                this->dq_dl[p][d] /= -3.0*this->cb->get_lx(d)*M*M/this->molecules->get_ds();
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
template <typename T>
void CudaComputationContinuous<T>::get_chain_propagator(T *q_out, int polymer, int v, int u, int n)
{
    // This method should be invoked after invoking compute_statistics()

    // Get chain propagator for a selected polymer, block and direction.
    // This is made for debugging and testing.
    try
    {
        const int M = this->cb->get_total_grid();
        Polymer& pc = this->molecules->get_polymer(polymer);
        std::string dep = pc.get_propagator_key(v,u);

        if (this->propagator_computation_optimizer->get_computation_propagators().find(dep) == this->propagator_computation_optimizer->get_computation_propagators().end())
            throw_with_line_number("Could not find the propagator code '" + dep + "'. Disable 'aggregation' option to obtain propagator_computation_optimizer.");

        const int N_RIGHT = this->propagator_computation_optimizer->get_computation_propagator(dep).max_n_segment;
        if (n < 0 || n > N_RIGHT)
            throw_with_line_number("n (" + std::to_string(n) + ") must be in range [0, " + std::to_string(N_RIGHT) + "]");

        gpu_error_check(hipMemcpy(q_out, d_propagator[dep][n], sizeof(T)*M, hipMemcpyDeviceToHost));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
template <typename T>
bool CudaComputationContinuous<T>::check_total_partition()
{
    const int M = this->cb->get_total_grid();
    int n_polymer_types = this->molecules->get_n_polymer_types();
    std::vector<std::vector<T>> total_partitions;
    for(int p=0;p<n_polymer_types;p++)
    {
        std::vector<T> total_partitions_p;
        total_partitions.push_back(total_partitions_p);
    }
    for(const auto& block: d_phi_block)
    {
        const auto& key = block.first;
        int p                 = std::get<0>(key);
        std::string key_left  = std::get<1>(key);
        std::string key_right = std::get<2>(key);

        int n_segment_right = this->propagator_computation_optimizer->get_computation_block(key).n_segment_right;
        int n_segment_left  = this->propagator_computation_optimizer->get_computation_block(key).n_segment_left;
        int n_repeated      = this->propagator_computation_optimizer->get_computation_block(key).n_repeated;
        int n_propagators   = this->propagator_computation_optimizer->get_computation_block(key).v_u.size();

        #ifndef NDEBUG
        std::cout<< p << ", " << key_left << ", " << key_right << ": " << n_segment_left << ", " << n_segment_right << ", " << n_propagators << ", " << this->propagator_computation_optimizer->get_computation_block(key).n_repeated << std::endl;
        #endif

        for(int n=0;n<=n_segment_right;n++)
        {
            T total_partition = dynamic_cast<CudaComputationBox<T>*>(this->cb)->inner_product_device(
                d_propagator[key_left][n_segment_left-n],
                d_propagator[key_right][n]);

            total_partition *= n_repeated/this->cb->get_volume()/n_propagators;
            total_partitions[p].push_back(total_partition);

            #ifndef NDEBUG
            std::cout<< p << ", " << n << ": " << total_partition << std::endl;
            #endif
        }
    }

    // Find minimum and maximum of total_partitions
    std::cout<< "Polymer id: maximum,  minimum, and difference of total partitions" << std::endl;
    for(size_t p=0;p<total_partitions.size();p++)
    {
        double max_partition = -1e20;
        double min_partition =  1e20;
        for(size_t n=0;n<total_partitions[p].size();n++)
        {
            if (std::abs(total_partitions[p][n]) > max_partition)
                max_partition = std::abs(total_partitions[p][n]);
            if (std::abs(total_partitions[p][n]) < min_partition)
                min_partition = std::abs(total_partitions[p][n]);
        }
        double diff_partition = std::abs(max_partition - min_partition);

        std::cout<< "\t" << p << ": " << max_partition << ", " << min_partition << ", " << diff_partition << std::endl;
        if (diff_partition > 1e-7)
            return false;
    }
    return true;
}

// Explicit template instantiation
template class CudaComputationContinuous<double>;
template class CudaComputationContinuous<std::complex<double>>;