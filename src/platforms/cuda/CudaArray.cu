
#include <iostream>
#include "CudaCommon.h"
#include "CudaArray.h"

CudaArray::CudaArray(unsigned new_size)
{
    gpu_error_check(hipMalloc((void**)&this->d_data, sizeof(double)*new_size));
    this->size = new_size;
    this->device="cuda";

    int device_id;
    gpu_error_check(hipGetDevice(&device_id));
    this->device_id= device_id;
}
CudaArray::CudaArray(double* new_data, unsigned int new_size) : CudaArray(new_size)
{
    gpu_error_check(hipMemcpy(d_data, new_data, sizeof(double)*new_size, hipMemcpyHostToDevice));
}
CudaArray::CudaArray(const Array& array) : CudaArray(array.get_size())
{
    unsigned int new_size = array.get_size();
    double *d_new_data = array.get_ptr();
    gpu_error_check(hipMemcpy(d_data, d_new_data, sizeof(double)*new_size, hipMemcpyDeviceToDevice));
}
CudaArray::~CudaArray()
{
    if (this->d_data != nullptr)
    {
        hipFree(this->d_data);
        this->size = 0;
        // std::cout << "Memory has been deallocated" << std::endl;
    }
}
void CudaArray::operator=(const Array& arr)
{
    unsigned int arr_size = arr.get_size();
    double *d_arr_data = arr.get_ptr();

    if (this->size != arr_size)
    {
        throw_with_line_number("Sizes of arrays ("
            + std::to_string(this->size) + ", "
            + std::to_string(arr_size) + ") do not match.");
    }
    gpu_error_check(hipMemcpy(d_data, d_arr_data, sizeof(double)*arr_size, hipMemcpyDeviceToDevice));
}
void CudaArray::set_data(double * arr_data, unsigned int arr_size)
{
    if (this->size != arr_size)
    {
        throw_with_line_number("Sizes of arrays ("
            + std::to_string(this->size) + ", "
            + std::to_string(arr_size) + ") do not match.");
    }
    gpu_error_check(hipMemcpy(d_data, arr_data, sizeof(double)*arr_size, hipMemcpyHostToDevice));
}
std::vector<double> CudaArray::to_vector() const
{
    double temp_arr[size];
    gpu_error_check(hipMemcpy(temp_arr, d_data, sizeof(double)*size, hipMemcpyDeviceToHost));

    std::vector<double> vec(temp_arr, temp_arr + size); 
    return vec;
}
double* CudaArray::get_ptr() const
{
    return this->d_data;
}
unsigned int CudaArray::get_size() const
{
    return this->size;
}
double CudaArray::operator[](unsigned int i) const
{
    if (size <= i)
        throw_with_line_number("Index [" + std::to_string(i) + "] is out of bound.");
    double element;
    gpu_error_check(hipMemcpy(&element, &d_data[i], sizeof(double), hipMemcpyDeviceToHost));
    return element;
}

// Arithmetic operations with two arrays
void CudaArray::add(const Array& src_1, const Array& src_2)
{
    unsigned int src1_size = src_1.get_size();
    unsigned int src2_size = src_2.get_size();
    double *d_src1_data = src_1.get_ptr();
    double *d_src2_data = src_2.get_ptr();

    if (this->size != src1_size || this->size != src2_size)
    {
        throw_with_line_number("Sizes of arrays ("
            + std::to_string(this->size) + ", "
            + std::to_string(src1_size)  + ", "
            + std::to_string(src2_size)  + ") do not match.");
    }

    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    int device_id;
    gpu_error_check(hipGetDevice(&device_id));
    if (this->device_id != device_id)
        throw_with_line_number("Device id does not match.");

    ker_lin_comb<double><<<N_BLOCKS, N_THREADS>>>(this->d_data, 1.0, d_src1_data, 1.0, d_src2_data, this->size);
}
void CudaArray::subtract(const Array& src_1, const Array& src_2)
{
    unsigned int src1_size = src_1.get_size();
    unsigned int src2_size = src_2.get_size();
    double *d_src1_data = src_1.get_ptr();
    double *d_src2_data = src_2.get_ptr();

    if (this->size != src1_size || this->size != src2_size)
    {
        throw_with_line_number("Sizes of arrays ("
            + std::to_string(this->size) + ", "
            + std::to_string(src1_size)  + ", "
            + std::to_string(src2_size)  + ") do not match.");
    }

    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    int device_id;
    gpu_error_check(hipGetDevice(&device_id));
    if (this->device_id != device_id)
        throw_with_line_number("Device id does not match.");

    ker_lin_comb<double><<<N_BLOCKS, N_THREADS>>>(this->d_data, 1.0, d_src1_data, -1.0, d_src2_data, this->size);
}
void CudaArray::multiply(const Array& src_1, const Array& src_2)
{
    unsigned int src1_size = src_1.get_size();
    unsigned int src2_size = src_2.get_size();
    double *d_src1_data = src_1.get_ptr();
    double *d_src2_data = src_2.get_ptr();

    if (this->size != src1_size || this->size != src2_size)
    {
        throw_with_line_number("Sizes of arrays ("
            + std::to_string(this->size) + ", "
            + std::to_string(src1_size)  + ", "
            + std::to_string(src2_size)  + ") do not match.");
    }

    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    int device_id;
    gpu_error_check(hipGetDevice(&device_id));
    if (this->device_id != device_id)
        throw_with_line_number("Device id does not match.");

    ker_multi<double><<<N_BLOCKS, N_THREADS>>>(this->d_data, d_src1_data, d_src2_data, 1.0, this->size);
}
void CudaArray::divide(const Array& src_1, const Array& src_2)
{
    unsigned int src1_size = src_1.get_size();
    unsigned int src2_size = src_2.get_size();
    double *d_src1_data = src_1.get_ptr();
    double *d_src2_data = src_2.get_ptr();

    if (this->size != src1_size || this->size != src2_size)
    {
        throw_with_line_number("Sizes of arrays ("
            + std::to_string(this->size) + ", "
            + std::to_string(src1_size)  + ", "
            + std::to_string(src2_size)  + ") do not match.");
    }

    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    int device_id;
    gpu_error_check(hipGetDevice(&device_id));
    if (this->device_id != device_id)
        throw_with_line_number("Device id does not match.");

    ker_divide<double><<<N_BLOCKS, N_THREADS>>>(this->d_data, d_src1_data, d_src2_data, 1.0, this->size);
}
// Arithmetic operations with an array and a float number
void CudaArray::linear_scaling(const Array& src, const double a, const double b)
{
    unsigned int src_size = src.get_size();
    double *d_src_data = src.get_ptr();

    if (this->size != src_size)
    {
        throw_with_line_number("Sizes of arrays ("
            + std::to_string(this->size) + ", "
            + std::to_string(src_size) + ") do not match.");
    }

    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    int device_id;
    gpu_error_check(hipGetDevice(&device_id));
    if (this->device_id != device_id)
        throw_with_line_number("Device id does not match.");

    ker_linear_scaling<double><<<N_BLOCKS, N_THREADS>>>(this->d_data, d_src_data, a, b, this->size);
}